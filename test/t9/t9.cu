#include <hip/hip_runtime.h>
#include <stdio.h>

// Simple CUDA kernel
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    // Error handling
    hipError_t err = hipSuccess;
    
    // Vector size
    const int N = 1024;
    size_t size = N * sizeof(float);
    
    // Host vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    
    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    
    // Create CUDA memory pool
    hipMemPool_t memPool;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0; // Current device
    poolProps.location.type = hipMemLocationTypeDevice;
    
    err = hipMemPoolCreate(&memPool, &poolProps);
    if (err != hipSuccess) {

        fprintf(stderr, "hipMemPoolCreate failed: %d %s\n", err, hipGetErrorString(err));
        return 1;
    }
    
    // Device vectors
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;
    
    // Allocate memory from pool
    err = hipMallocFromPoolAsync(&d_A, size, memPool, 0);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocFromPoolAsync failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    err = hipMallocFromPoolAsync(&d_B, size, memPool, 0);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocFromPoolAsync failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    err = hipMallocFromPoolAsync(&d_C, size, memPool, 0);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMallocFromPoolAsync failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Copy inputs to device
    err = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpyAsync failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    err = hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, 0);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpyAsync failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, 0>>>(d_A, d_B, d_C, N);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Copy result back
    err = hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, 0);
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpyAsync failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Synchronize
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Verify result
    for (int i = 0; i < N; i++) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            return 1;
        }
    }
    
    printf("Vector addition completed successfully!\n");
    
    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipMemPoolDestroy(memPool);
    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
}
