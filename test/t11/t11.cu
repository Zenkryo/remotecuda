#include <hip/hip_runtime.h>
#include <stdio.h>

// Simple CUDA kernel to initialize array
__global__ void initKernel(int *data, int value, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N) {
        data[idx] = value;
    }
}

int main() {
// Error checking macro
#define CUDA_CHECK(call)                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                       \
    {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                          \
        hipError_t err = call;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                \
        if(err != hipSuccess) {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                               \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));                                                                                                                                                                                                                                                                                                                                                                                                                         \
            exit(1);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                           \
        }                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                      \
    }

    // Initialize data
    const int N = 1024;
    size_t size = N * sizeof(int);

    // Allocate host memory for verification
    int *h_data = (int *)malloc(size);
    if(!h_data) {
        fprintf(stderr, "Host memory allocation failed\n");
        exit(1);
    }

    // Allocate device memory
    int *d_data;
    CUDA_CHECK(hipMalloc(&d_data, size));

    // Create CUDA graph
    hipGraph_t graph;
    CUDA_CHECK(hipGraphCreate(&graph, 0));

    // Create kernel node parameters
    hipKernelNodeParams kernelNodeParams = {0};
    int value = 42; // Value to initialize array
    void *kernelArgs[] = {&d_data, (void *)&value, (void *)&N};

    kernelNodeParams.func = (void *)initKernel;
    kernelNodeParams.gridDim = dim3((N + 255) / 256, 1, 1);
    kernelNodeParams.blockDim = dim3(256, 1, 1);
    kernelNodeParams.sharedMemBytes = 0;
    kernelNodeParams.kernelParams = kernelArgs;
    kernelNodeParams.extra = NULL;

    // Add kernel node to graph
    hipGraphNode_t kernelNode;
    CUDA_CHECK(hipGraphAddKernelNode(&kernelNode, graph, NULL, 0, &kernelNodeParams));

    // Create a stream for the graph
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Instantiate and launch graph
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Launch the graph
    CUDA_CHECK(hipGraphLaunch(graphExec, stream));

    // Wait for graph execution to complete
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Free the device memory after graph execution
    CUDA_CHECK(hipFree(d_data));

    // Verify memory is freed by attempting to access it
    hipError_t err = hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    if(err == hipErrorInvalidValue || err == hipErrorInvalidDevicePointer) {
        printf("Memory successfully freed\n");
    } else {
        printf("Unexpected error or memory not freed: %s\n", hipGetErrorString(err));
    }

    // Cleanup
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipGraphExecDestroy(graphExec));
    CUDA_CHECK(hipGraphDestroy(graph));
    free(h_data);

    return 0;
}
