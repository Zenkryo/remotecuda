#include <stdio.h>
#include <hip/hip_runtime.h>

#define DRIVER_API_CALL(apiFuncCall)                                   \
do {                                                                   \
    hipError_t _status = apiFuncCall;                                    \
    if (_status != hipSuccess) {                                     \
        const char *errName;                                           \
        hipDrvGetErrorName(_status, &errName);                            \
        fprintf(stderr, "CUDA driver API error %d: %s\n", _status, errName); \
        exit(1);                                                       \
    }                                                                  \
} while (0)

int main() {
    // 初始化CUDA驱动API
    DRIVER_API_CALL(hipInit(0));

    // 获取设备
    hipDevice_t device;
    DRIVER_API_CALL(hipDeviceGet(&device, 0));

    // 创建上下文
    hipCtx_t context;
    DRIVER_API_CALL(hipCtxCreate(&context, 0, device));

    // 设置虚拟内存分配参数
    size_t allocationSize = 1 << 24; // 16MB
    size_t alignment = 1 << 16;     // 64KB对齐

    // 保留虚拟地址范围
    hipDeviceptr_t ptr;
    DRIVER_API_CALL(hipMemAddressReserve(&ptr, allocationSize, alignment, 0, 0));

    printf("Reserved virtual address range: 0x%llx - 0x%llx (%zu bytes)\n",
           (unsigned long long)ptr, 
           (unsigned long long)ptr + allocationSize - 1,
           allocationSize);

    // 分配物理内存
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;

    hipMemGenericAllocationHandle_t handle;
    DRIVER_API_CALL(hipMemCreate(&handle, allocationSize, &prop, 0));

    // 将物理内存映射到保留的虚拟地址范围
    DRIVER_API_CALL(hipMemMap(ptr, allocationSize, 0, handle, 0));

    // 设置访问权限
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = device;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;

    DRIVER_API_CALL(hipMemSetAccess(ptr, allocationSize, &accessDesc, 1));

    // 使用分配的内存（示例：填充数据）
    int *d_data = (int*)ptr;
    int numElements = allocationSize / sizeof(int);
    
    // 启动核函数填充数据（这里简化处理，实际使用时需要编写核函数）
    printf("Memory successfully reserved and mapped. Ready for use.\n");

    // 清理资源
    DRIVER_API_CALL(hipMemUnmap(ptr, allocationSize));
    DRIVER_API_CALL(hipMemRelease(handle));
    DRIVER_API_CALL(hipMemAddressFree(ptr, allocationSize));
    DRIVER_API_CALL(hipCtxDestroy(context));

    return 0;
}
