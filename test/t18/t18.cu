#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main() {
    hipStream_t stream;
    hipError_t err;

    int leastPriority, greatestPriority;
    err = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);

    err = hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, greatestPriority);

    // Test cudaStreamGetPriority
    int priority;
    err = hipStreamGetPriority(stream, &priority);

    // Test cudaStreamGetFlags
    unsigned int flags;
    err = hipStreamGetFlags(stream, &flags);

    err = hipStreamDestroy(stream);
}
