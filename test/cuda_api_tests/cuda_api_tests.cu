#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

// 定义设备端的全局变量
__device__ int dev_data;

// 简单的核函数，用于测试
__global__ void test_kernel() {
    // 空核函数，仅用于测试启动
}

class CudaApiTest : public ::testing::Test {
  protected:
    hipDevice_t device;
    hipCtx_t context;

    void SetUp() override {
        hipError_t result = hipInit(0);
        if(result != hipSuccess) {
            throw std::runtime_error("Failed to initialize CUDA");
        }

        result = hipDeviceGet(&device, 0);
        if(result != hipSuccess) {
            throw std::runtime_error("Failed to get CUDA device");
        }

        result = hipCtxCreate(&context, 0, device);
        if(result != hipSuccess) {
            throw std::runtime_error("Failed to create CUDA context");
        }
    }

    void TearDown() override {
        if(context) {
            hipCtxDestroy(context);
        }
    }
};

// 辅助函数用于检查CUDA错误
void checkCudaError(hipError_t error, const char *message, const char *file, int line) {
    if(error != hipSuccess) {
        const char *errorName = hipGetErrorName(error);
        const char *errorString = hipGetErrorString(error);
        FAIL() << "Error at " << file << ":" << line << " - " << message << ": " << errorName << " - " << errorString;
    }
}

void checkCuError(hipError_t result, const char *message, const char *file, int line) {
    if(result != hipSuccess) {
        const char *errorName;
        hipDrvGetErrorName(result, &errorName);
        const char *errorString;
        hipDrvGetErrorString(result, &errorString);
        FAIL() << "Error at " << file << ":" << line << " - " << message << ": " << errorName << " - " << errorString;
    }
}

// 宏定义用于简化错误检查调用
#define CHECK_CUDA_ERROR(err, msg) checkCudaError(err, msg, __FILE__, __LINE__)
#define CHECK_CU_ERROR(result, msg) checkCuError(result, msg, __FILE__, __LINE__)

// Test hipFree
TEST_F(CudaApiTest, CudaFree) {
    void *devPtr;
    hipError_t err = hipMalloc(&devPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    ASSERT_NE(devPtr, nullptr);

    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
}

// Test hipHostFree
TEST_F(CudaApiTest, CudaFreeHost) {
    void *hostPtr;
    hipError_t err = hipHostMalloc(&hostPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate host memory");
    ASSERT_NE(hostPtr, nullptr);

    err = hipHostFree(hostPtr);
    CHECK_CUDA_ERROR(err, "Failed to free host memory");
}

// Test hipGetErrorName and hipGetErrorString
TEST_F(CudaApiTest, CudaGetErrorInfo) {
    const char *errorName = hipGetErrorName(hipSuccess);
    ASSERT_NE(errorName, nullptr);
    ASSERT_STREQ(errorName, "hipSuccess");

    const char *errorString = hipGetErrorString(hipSuccess);
    ASSERT_NE(errorString, nullptr);
    ASSERT_STREQ(errorString, "no error");
}

// Test hipGetSymbolAddress
TEST_F(CudaApiTest, CudaGetSymbolAddress) {
    void *devPtr;
    hipError_t err = hipGetSymbolAddress(&devPtr, HIP_SYMBOL(dev_data));
    CHECK_CUDA_ERROR(err, "Failed to get symbol address");
    ASSERT_NE(devPtr, nullptr);

    // Write a value to the device variable
    int testValue = 42;
    err = hipMemcpy(devPtr, &testValue, sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err, "Failed to write to device variable");

    // Read back the value
    int readValue = 0;
    err = hipMemcpy(&readValue, devPtr, sizeof(int), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err, "Failed to read from device variable");

    // Verify the value
    ASSERT_EQ(readValue, testValue);

    // Test with a different value
    testValue = 123;
    err = hipMemcpy(devPtr, &testValue, sizeof(int), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err, "Failed to write second value to device variable");

    err = hipMemcpy(&readValue, devPtr, sizeof(int), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err, "Failed to read second value from device variable");

    ASSERT_EQ(readValue, testValue);
}

// Test hipHostAlloc with different flags
TEST_F(CudaApiTest, CudaHostAlloc) {
    struct TestCase {
        unsigned int flags;
        const char *description;
    };

    TestCase testCases[] = {{hipHostMallocDefault, "Default flags"}, {hipHostMallocPortable, "Portable memory"}, {hipHostMallocMapped, "Mapped memory"}, {hipHostMallocWriteCombined, "Write-combined memory"}};

    for(const auto &testCase : testCases) {
        void *hostPtr;
        hipError_t err = hipHostAlloc(&hostPtr, 1024, testCase.flags);
        CHECK_CUDA_ERROR(err, (std::string("Failed to allocate host memory with ") + testCase.description).c_str());
        ASSERT_NE(hostPtr, nullptr);

        // Test writing to host memory
        int *intPtr = static_cast<int *>(hostPtr);
        for(int i = 0; i < 256; i++) { // 1024 bytes / sizeof(int) = 256 ints
            intPtr[i] = i;
        }

        // Test reading from host memory
        for(int i = 0; i < 256; i++) {
            ASSERT_EQ(intPtr[i], i) << "Memory verification failed at index " << i << " with flags " << testCase.description;
        }

        // Test writing a different pattern
        for(int i = 0; i < 256; i++) {
            intPtr[i] = 255 - i;
        }

        // Verify the new pattern
        for(int i = 0; i < 256; i++) {
            ASSERT_EQ(intPtr[i], 255 - i) << "Memory verification failed at index " << i << " with flags " << testCase.description;
        }

        err = hipHostFree(hostPtr);
        CHECK_CUDA_ERROR(err, (std::string("Failed to free host memory with ") + testCase.description).c_str());
    }
}

// Test hipHostRegister and hipHostUnregister with different flags
TEST_F(CudaApiTest, CudaHostRegister) {
    struct TestCase {
        unsigned int flags;
        const char *description;
        bool optional; // 标记是否为可选功能
    };

    TestCase testCases[] = {
        {hipHostRegisterDefault, "Default flags", false}, {hipHostRegisterPortable, "Portable memory", false}, {hipHostRegisterMapped, "Mapped memory", false}, {hipHostRegisterIoMemory, "I/O memory", true} // 标记为可选功能
    };

    for(const auto &testCase : testCases) {
        void *hostPtr = malloc(1024);
        ASSERT_NE(hostPtr, nullptr);

        hipError_t err = hipHostRegister(hostPtr, 1024, testCase.flags);

        if(testCase.optional) {
            // 对于可选功能，如果返回不支持的错误，我们认为是正常的
            if(err == hipErrorInvalidValue || err == hipErrorNotSupported) {
                // 跳过这个测试用例
                free(hostPtr);
                continue;
            }
        }

        CHECK_CUDA_ERROR(err, (std::string("Failed to register host memory with ") + testCase.description).c_str());

        err = hipHostUnregister(hostPtr);
        CHECK_CUDA_ERROR(err, (std::string("Failed to unregister host memory with ") + testCase.description).c_str());

        free(hostPtr);
    }
}

// Test hipLaunchKernel with different grid and block dimensions
TEST_F(CudaApiTest, CudaLaunchKernel) {
    struct TestCase {
        dim3 gridDim;
        dim3 blockDim;
        const char *description;
    };

    TestCase testCases[] = {{dim3(1, 1, 1), dim3(1, 1, 1), "1x1x1 grid and block"}, {dim3(2, 2, 1), dim3(32, 32, 1), "2x2x1 grid and 32x32x1 block"}, {dim3(4, 4, 1), dim3(16, 16, 1), "4x4x1 grid and 16x16x1 block"}};

    for(const auto &testCase : testCases) {
        void *args[] = {};
        hipError_t err = hipLaunchKernel((const void *)test_kernel, testCase.gridDim, testCase.blockDim, args, 0, 0);
        CHECK_CUDA_ERROR(err, (std::string("Failed to launch kernel with ") + testCase.description).c_str());

        err = hipDeviceSynchronize();
        CHECK_CUDA_ERROR(err, (std::string("Failed to synchronize after kernel launch with ") + testCase.description).c_str());
    }
}

// Test hipMalloc with different sizes
TEST_F(CudaApiTest, CudaMalloc) {
    size_t sizes[] = {1, 1024, 1024 * 1024}; // Remove 1GB test

    for(size_t size : sizes) {
        void *devPtr;
        hipError_t err = hipMalloc(&devPtr, size);
        CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
        ASSERT_NE(devPtr, nullptr);

        // Allocate host memory for testing
        void *hostPtr = malloc(size);
        ASSERT_NE(hostPtr, nullptr);

        // Initialize host memory with a pattern
        int *hostIntPtr = static_cast<int *>(hostPtr);
        size_t numInts = size / sizeof(int);
        for(size_t i = 0; i < numInts; i++) {
            hostIntPtr[i] = static_cast<int>(i);
        }

        // Copy from host to device
        err = hipMemcpy(devPtr, hostPtr, size, hipMemcpyHostToDevice);
        CHECK_CUDA_ERROR(err, "Failed to copy from host to device");

        // Clear host memory
        memset(hostPtr, 0, size);

        // Copy back from device to host
        err = hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);
        CHECK_CUDA_ERROR(err, "Failed to copy from device to host");

        // Verify the data
        for(size_t i = 0; i < numInts; i++) {
            ASSERT_EQ(hostIntPtr[i], static_cast<int>(i)) << "Memory verification failed at index " << i << " with size " << size;
        }

        // Test with a different pattern
        for(size_t i = 0; i < numInts; i++) {
            hostIntPtr[i] = static_cast<int>(numInts - 1 - i);
        }

        // Copy the new pattern to device
        err = hipMemcpy(devPtr, hostPtr, size, hipMemcpyHostToDevice);
        CHECK_CUDA_ERROR(err, "Failed to copy second pattern to device");

        // Clear host memory again
        memset(hostPtr, 0, size);

        // Copy back from device to host
        err = hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);
        CHECK_CUDA_ERROR(err, "Failed to copy second pattern from device");

        // Verify the new pattern
        for(size_t i = 0; i < numInts; i++) {
            ASSERT_EQ(hostIntPtr[i], static_cast<int>(numInts - 1 - i)) << "Second pattern verification failed at index " << i << " with size " << size;
        }

        // Clean up
        free(hostPtr);
        err = hipFree(devPtr);
        CHECK_CUDA_ERROR(err, "Failed to free device memory");
    }
}

// Test hipMalloc3D with different extents
TEST_F(CudaApiTest, CudaMalloc3D) {
    struct TestCase {
        hipExtent extent;
        const char *description;
    };

    TestCase testCases[] = {{make_hipExtent(32, 32, 1), "32x32x1"}, {make_hipExtent(64, 64, 1), "64x64x1"}, {make_hipExtent(128, 128, 1), "128x128x1"}};

    for(const auto &testCase : testCases) {
        hipPitchedPtr pitchedDevPtr;
        hipError_t err = hipMalloc3D(&pitchedDevPtr, testCase.extent);
        CHECK_CUDA_ERROR(err, (std::string("Failed to allocate 3D memory with extent ") + testCase.description).c_str());
        ASSERT_NE(pitchedDevPtr.ptr, nullptr);

        // // Create host memory for testing
        // size_t hostPitch = testCase.extent.width * sizeof(int);
        // size_t hostSize = hostPitch * testCase.extent.height * testCase.extent.depth;
        // void *hostPtr = malloc(hostSize);
        // ASSERT_NE(hostPtr, nullptr);

        // // Initialize host memory with a pattern
        // int *hostIntPtr = static_cast<int *>(hostPtr);
        // for(size_t z = 0; z < testCase.extent.depth; z++) {
        //     for(size_t y = 0; y < testCase.extent.height; y++) {
        //         for(size_t x = 0; x < testCase.extent.width; x++) {
        //             size_t index = z * testCase.extent.height * testCase.extent.width + y * testCase.extent.width + x;
        //             hostIntPtr[index] = static_cast<int>(index);
        //         }
        //     }
        // }

        // // Copy from host to device
        // hipMemcpy3DParms copyParams = {0};
        // copyParams.srcPtr = make_hipPitchedPtr(hostPtr, hostPitch, testCase.extent.width, testCase.extent.height);
        // copyParams.dstPtr = pitchedDevPtr;
        // copyParams.extent = testCase.extent;
        // copyParams.kind = hipMemcpyHostToDevice;

        // err = hipMemcpy3D(&copyParams);
        // CHECK_CUDA_ERROR(err, (std::string("Failed to copy from host to device with extent ") + testCase.description).c_str());

        // // Clear host memory
        // memset(hostPtr, 0, hostSize);

        // // Copy back from device to host
        // copyParams.srcPtr = pitchedDevPtr;
        // copyParams.dstPtr = make_hipPitchedPtr(hostPtr, hostPitch, testCase.extent.width, testCase.extent.height);
        // copyParams.kind = hipMemcpyDeviceToHost;

        // err = hipMemcpy3D(&copyParams);
        // CHECK_CUDA_ERROR(err, (std::string("Failed to copy from device to host with extent ") + testCase.description).c_str());

        // // Verify the data
        // for(size_t z = 0; z < testCase.extent.depth; z++) {
        //     for(size_t y = 0; y < testCase.extent.height; y++) {
        //         for(size_t x = 0; x < testCase.extent.width; x++) {
        //             size_t index = z * testCase.extent.height * testCase.extent.width + y * testCase.extent.width + x;
        //             ASSERT_EQ(hostIntPtr[index], static_cast<int>(index)) << "Memory verification failed at position (" << x << "," << y << "," << z << ") with extent " << testCase.description;
        //         }
        //     }
        // }

        // // Test with a different pattern
        // for(size_t z = 0; z < testCase.extent.depth; z++) {
        //     for(size_t y = 0; y < testCase.extent.height; y++) {
        //         for(size_t x = 0; x < testCase.extent.width; x++) {
        //             size_t index = z * testCase.extent.height * testCase.extent.width + y * testCase.extent.width + x;
        //             hostIntPtr[index] = static_cast<int>(testCase.extent.width * testCase.extent.height * testCase.extent.depth - 1 - index);
        //         }
        //     }
        // }

        // // Copy the new pattern to device
        // copyParams.srcPtr = make_hipPitchedPtr(hostPtr, hostPitch, testCase.extent.width, testCase.extent.height);
        // copyParams.dstPtr = pitchedDevPtr;
        // copyParams.kind = hipMemcpyHostToDevice;

        // err = hipMemcpy3D(&copyParams);
        // CHECK_CUDA_ERROR(err, (std::string("Failed to copy second pattern to device with extent ") + testCase.description).c_str());

        // // Clear host memory again
        // memset(hostPtr, 0, hostSize);

        // // Copy back from device to host
        // copyParams.srcPtr = pitchedDevPtr;
        // copyParams.dstPtr = make_hipPitchedPtr(hostPtr, hostPitch, testCase.extent.width, testCase.extent.height);
        // copyParams.kind = hipMemcpyDeviceToHost;

        // err = hipMemcpy3D(&copyParams);
        // CHECK_CUDA_ERROR(err, (std::string("Failed to copy second pattern from device with extent ") + testCase.description).c_str());

        // // Verify the new pattern
        // for(size_t z = 0; z < testCase.extent.depth; z++) {
        //     for(size_t y = 0; y < testCase.extent.height; y++) {
        //         for(size_t x = 0; x < testCase.extent.width; x++) {
        //             size_t index = z * testCase.extent.height * testCase.extent.width + y * testCase.extent.width + x;
        //             ASSERT_EQ(hostIntPtr[index], static_cast<int>(testCase.extent.width * testCase.extent.height * testCase.extent.depth - 1 - index)) << "Second pattern verification failed at position (" << x << "," << y << "," << z << ") with extent " << testCase.description;
        //         }
        //     }
        // }

        // // Clean up
        // free(hostPtr);
        err = hipFree(pitchedDevPtr.ptr);
        CHECK_CUDA_ERROR(err, (std::string("Failed to free 3D memory with extent ") + testCase.description).c_str());
    }
}

// Test hipHostMalloc
TEST_F(CudaApiTest, CudaMallocHost) {
    void *hostPtr;
    hipError_t err = hipHostMalloc(&hostPtr, 1024);
    ASSERT_EQ(err, hipSuccess);
    ASSERT_NE(hostPtr, nullptr);

    err = hipHostFree(hostPtr);
    ASSERT_EQ(err, hipSuccess);
}

// Test hipMallocManaged
TEST_F(CudaApiTest, CudaMallocManaged) {
    void *devPtr;
    hipError_t err = hipMallocManaged(&devPtr, 1024, hipMemAttachGlobal);
    ASSERT_EQ(err, hipSuccess);
    ASSERT_NE(devPtr, nullptr);

    // Test writing to managed memory
    int *intPtr = static_cast<int *>(devPtr);
    for(int i = 0; i < 256; i++) { // 1024 bytes / sizeof(int) = 256 ints
        intPtr[i] = i;
    }

    // Test reading from managed memory
    for(int i = 0; i < 256; i++) {
        ASSERT_EQ(intPtr[i], i) << "Managed memory verification failed at index " << i;
    }

    // Test with a different pattern
    for(int i = 0; i < 256; i++) {
        intPtr[i] = 255 - i;
    }

    // Verify the new pattern
    for(int i = 0; i < 256; i++) {
        ASSERT_EQ(intPtr[i], 255 - i) << "Second pattern verification failed at index " << i;
    }

    err = hipFree(devPtr);
    ASSERT_EQ(err, hipSuccess);
}

// Test hipMallocPitch
TEST_F(CudaApiTest, CudaMallocPitch) {
    void *devPtr;
    size_t pitch;
    hipError_t err = hipMallocPitch(&devPtr, &pitch, 32, 32);
    ASSERT_EQ(err, hipSuccess);
    ASSERT_NE(devPtr, nullptr);
    ASSERT_GT(pitch, 0);

    // Allocate host memory with the same pitch
    void *hostPtr = malloc(pitch * 32);
    ASSERT_NE(hostPtr, nullptr);

    // Initialize host memory with a pattern
    int *hostIntPtr = static_cast<int *>(hostPtr);
    for(int y = 0; y < 32; y++) {
        for(int x = 0; x < 32; x++) {
            hostIntPtr[y * (pitch / sizeof(int)) + x] = y * 32 + x;
        }
    }

    // Copy from host to device
    err = hipMemcpy2D(devPtr, pitch, hostPtr, pitch, 32 * sizeof(int), 32, hipMemcpyHostToDevice);
    ASSERT_EQ(err, hipSuccess);

    // Clear host memory
    memset(hostPtr, 0, pitch * 32);

    // Copy back from device to host
    err = hipMemcpy2D(hostPtr, pitch, devPtr, pitch, 32 * sizeof(int), 32, hipMemcpyDeviceToHost);
    ASSERT_EQ(err, hipSuccess);

    // Verify the data
    for(int y = 0; y < 32; y++) {
        for(int x = 0; x < 32; x++) {
            ASSERT_EQ(hostIntPtr[y * (pitch / sizeof(int)) + x], y * 32 + x) << "Memory verification failed at position (" << x << "," << y << ")";
        }
    }

    // Test with a different pattern
    for(int y = 0; y < 32; y++) {
        for(int x = 0; x < 32; x++) {
            hostIntPtr[y * (pitch / sizeof(int)) + x] = (31 - y) * 32 + (31 - x);
        }
    }

    // Copy the new pattern to device
    err = hipMemcpy2D(devPtr, pitch, hostPtr, pitch, 32 * sizeof(int), 32, hipMemcpyHostToDevice);
    ASSERT_EQ(err, hipSuccess);

    // Clear host memory again
    memset(hostPtr, 0, pitch * 32);

    // Copy back from device to host
    err = hipMemcpy2D(hostPtr, pitch, devPtr, pitch, 32 * sizeof(int), 32, hipMemcpyDeviceToHost);
    ASSERT_EQ(err, hipSuccess);

    // Verify the new pattern
    for(int y = 0; y < 32; y++) {
        for(int x = 0; x < 32; x++) {
            ASSERT_EQ(hostIntPtr[y * (pitch / sizeof(int)) + x], (31 - y) * 32 + (31 - x)) << "Second pattern verification failed at position (" << x << "," << y << ")";
        }
    }

    // Clean up
    free(hostPtr);
    err = hipFree(devPtr);
    ASSERT_EQ(err, hipSuccess);
}

// Test CUDA Driver API functions
TEST_F(CudaApiTest, CudaDriverApi) {
    hipError_t result;

    // Test hipDrvGetErrorName and hipDrvGetErrorString
    const char *errorName;
    result = hipDrvGetErrorName(hipSuccess, &errorName);
    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(errorName, nullptr);

    const char *errorString;
    result = hipDrvGetErrorString(hipSuccess, &errorString);
    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(errorString, nullptr);

    // Test hipMalloc
    hipDeviceptr_t devPtr;
    result = hipMalloc(&devPtr, 1024);
    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(devPtr, 0);

    // Allocate host memory for testing
    void *hostPtr = malloc(1024);
    ASSERT_NE(hostPtr, nullptr);

    // Initialize host memory with a pattern
    int *hostIntPtr = static_cast<int *>(hostPtr);
    for(int i = 0; i < 256; i++) { // 1024 bytes / sizeof(int) = 256 ints
        hostIntPtr[i] = i;
    }

    // Copy from host to device
    result = hipMemcpyHtoD(devPtr, hostPtr, 1024);
    ASSERT_EQ(result, hipSuccess);

    // Clear host memory
    memset(hostPtr, 0, 1024);

    // Copy back from device to host
    result = hipMemcpyDtoH(hostPtr, devPtr, 1024);
    ASSERT_EQ(result, hipSuccess);

    // Verify the data
    for(int i = 0; i < 256; i++) {
        ASSERT_EQ(hostIntPtr[i], i) << "Memory verification failed at index " << i;
    }

    // Test with a different pattern
    for(int i = 0; i < 256; i++) {
        hostIntPtr[i] = 255 - i;
    }

    // Copy the new pattern to device
    result = hipMemcpyHtoD(devPtr, hostPtr, 1024);
    ASSERT_EQ(result, hipSuccess);

    // Clear host memory again
    memset(hostPtr, 0, 1024);

    // Copy back from device to host
    result = hipMemcpyDtoH(hostPtr, devPtr, 1024);
    ASSERT_EQ(result, hipSuccess);

    // Verify the new pattern
    for(int i = 0; i < 256; i++) {
        ASSERT_EQ(hostIntPtr[i], 255 - i) << "Second pattern verification failed at index " << i;
    }

    // Test hipFree
    result = hipFree(devPtr);
    ASSERT_EQ(result, hipSuccess);

    // Test hipMemAllocHost
    void *hostPtr2;
    result = hipMemAllocHost(&hostPtr2, 1024);
    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(hostPtr2, nullptr);

    // Test writing to host memory
    int *hostIntPtr2 = static_cast<int *>(hostPtr2);
    for(int i = 0; i < 256; i++) {
        hostIntPtr2[i] = i;
    }

    // Test reading from host memory
    for(int i = 0; i < 256; i++) {
        ASSERT_EQ(hostIntPtr2[i], i) << "Host memory verification failed at index " << i;
    }

    // Test with a different pattern
    for(int i = 0; i < 256; i++) {
        hostIntPtr2[i] = 255 - i;
    }

    // Verify the new pattern
    for(int i = 0; i < 256; i++) {
        ASSERT_EQ(hostIntPtr2[i], 255 - i) << "Second host pattern verification failed at index " << i;
    }

    // Test hipHostFree
    result = hipHostFree(hostPtr2);
    ASSERT_EQ(result, hipSuccess);

    // Clean up
    free(hostPtr);
}

// Test hipExternalMemoryGetMappedBuffer
TEST_F(CudaApiTest, CuExternalMemoryGetMappedBuffer) {
    hipError_t result;
    hipDeviceptr_t devPtr = 0;
    hipExternalMemory_t extMem = nullptr;
    hipExternalMemoryBufferDesc bufferDesc = {};

    result = hipExternalMemoryGetMappedBuffer(&devPtr, extMem, &bufferDesc);
    // This test is expected to fail since we don't have a valid external memory handle
    ASSERT_NE(result, hipSuccess);
}

// Test hipGraphicsResourceGetMappedPointer
TEST_F(CudaApiTest, CuGraphicsResourceGetMappedPointer) {
    hipError_t result;
    hipDeviceptr_t devPtr;
    size_t size;
    hipGraphicsResource_t resource = nullptr;

    result = hipGraphicsResourceGetMappedPointer(&devPtr, &size, resource);
    // This test is expected to fail since we don't have a valid graphics resource
    ASSERT_NE(result, hipSuccess);
    ASSERT_EQ(result, hipErrorInvalidHandle);
}

// Test hipImportExternalMemory
TEST_F(CudaApiTest, CuImportExternalMemory) {
    hipError_t result;
    hipExternalMemory_t extMem;
    hipExternalMemoryHandleDesc memHandleDesc = {};

    result = hipImportExternalMemory(&extMem, &memHandleDesc);
    // This test is expected to fail since we don't have a valid memory handle
    ASSERT_NE(result, hipSuccess);
}

// Test cuIpcOpenMemHandle_v2
TEST_F(CudaApiTest, CuIpcOpenMemHandle) {
    hipError_t result;
    hipDeviceptr_t devPtr = 0;
    hipIpcMemHandle_t handle = {};
    unsigned int flags = 0;

    result = cuIpcOpenMemHandle_v2(&devPtr, handle, flags);
    // This test is expected to fail since we don't have a valid IPC memory handle
    ASSERT_NE(result, hipSuccess);
}

// Test hipModuleLaunchCooperativeKernel
TEST_F(CudaApiTest, CuLaunchCooperativeKernel) {
    hipError_t result;
    hipFunction_t func = nullptr;
    unsigned int gridDimX = 1, gridDimY = 1, gridDimZ = 1;
    unsigned int blockDimX = 1, blockDimY = 1, blockDimZ = 1;
    unsigned int sharedMemBytes = 0;
    hipStream_t hStream = 0;
    void **kernelParams = nullptr;

    result = hipModuleLaunchCooperativeKernel(func, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, hStream, kernelParams);
    // This test is expected to fail since we don't have a valid function handle
    ASSERT_NE(result, hipSuccess);
}

// Test hipMemAddressReserve
TEST_F(CudaApiTest, CuMemAddressReserve) {
    hipError_t result;
    hipDeviceptr_t ptr;

    // 设置虚拟内存分配参数
    size_t allocationSize = 1 << 24; // 16MB
    size_t alignment = 1 << 16;      // 64KB对齐

    // 保留虚拟地址范围
    result = hipMemAddressReserve(&ptr, allocationSize, alignment, 0, 0);

    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(ptr, 0);

    // Clean up
    result = hipMemAddressFree(ptr, allocationSize);
    ASSERT_EQ(result, hipSuccess);
}

// Test hipMemCreate
TEST_F(CudaApiTest, CuMemCreate) {
    hipError_t result;

    // 查询设备属性
    size_t granularity = 0;
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;

    hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);

    // 确保大小是粒度的整数倍
    size_t size = ((1024 * 1024) + granularity - 1) & ~(granularity - 1); // 1MB对齐

    // 创建内存分配
    hipMemGenericAllocationHandle_t handle;
    result = hipMemCreate(&handle, size, &prop, 0);
    ASSERT_EQ(result, hipSuccess);

    // 保留虚拟地址范围
    hipDeviceptr_t ptr;
    result = hipMemAddressReserve(&ptr, size, granularity, 0, 0);
    ASSERT_EQ(result, hipSuccess);

    // 映射内存
    result = hipMemMap(ptr, size, 0, handle, 0);
    ASSERT_EQ(result, hipSuccess);

    // 设置访问权限
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = device;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;

    result = hipMemSetAccess(ptr, size, &accessDesc, 1);
    ASSERT_EQ(result, hipSuccess);

    // Allocate host memory for testing
    void *hostPtr = malloc(size);
    ASSERT_NE(hostPtr, nullptr);

    // Initialize host memory with a pattern
    int *hostIntPtr = static_cast<int *>(hostPtr);
    size_t numInts = size / sizeof(int);
    for(size_t i = 0; i < numInts; i++) {
        hostIntPtr[i] = static_cast<int>(i);
    }

    // Copy from host to device
    result = hipMemcpyHtoD(ptr, hostPtr, size);
    ASSERT_EQ(result, hipSuccess);

    // Clear host memory
    memset(hostPtr, 0, size);

    // Copy back from device to host
    result = hipMemcpyDtoH(hostPtr, ptr, size);
    ASSERT_EQ(result, hipSuccess);

    // Verify the data
    for(size_t i = 0; i < numInts; i++) {
        ASSERT_EQ(hostIntPtr[i], static_cast<int>(i)) << "Memory verification failed at index " << i;
    }

    // Test with a different pattern
    for(size_t i = 0; i < numInts; i++) {
        hostIntPtr[i] = static_cast<int>(numInts - 1 - i);
    }

    // Copy the new pattern to device
    result = hipMemcpyHtoD(ptr, hostPtr, size);
    ASSERT_EQ(result, hipSuccess);

    // Clear host memory again
    memset(hostPtr, 0, size);

    // Copy back from device to host
    result = hipMemcpyDtoH(hostPtr, ptr, size);
    ASSERT_EQ(result, hipSuccess);

    // Verify the new pattern
    for(size_t i = 0; i < numInts; i++) {
        ASSERT_EQ(hostIntPtr[i], static_cast<int>(numInts - 1 - i)) << "Second pattern verification failed at index " << i;
    }

    // Clean up
    result = hipMemUnmap(ptr, size);
    ASSERT_EQ(result, hipSuccess);

    result = hipMemAddressFree(ptr, size);
    ASSERT_EQ(result, hipSuccess);

    result = hipMemRelease(handle);
    ASSERT_EQ(result, hipSuccess);

    free(hostPtr);
}

// Test hipMemGetAddressRange
TEST_F(CudaApiTest, CuMemGetAddressRange) {
    hipError_t result;
    hipDeviceptr_t base;
    size_t size;
    hipDeviceptr_t devPtr;

    // First allocate some memory
    result = hipMalloc(&devPtr, 1024);
    ASSERT_EQ(result, hipSuccess);

    result = hipMemGetAddressRange(&base, &size, devPtr);
    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(base, 0);
    ASSERT_GE(size, 1024);

    // Clean up
    result = hipFree(devPtr);
    ASSERT_EQ(result, hipSuccess);
}

// Test hipHostAlloc
TEST_F(CudaApiTest, CuMemHostAlloc) {
    hipError_t result;
    void *hostPtr;
    size_t bytesize = 1024;
    unsigned int flags = 0;

    result = hipHostAlloc(&hostPtr, bytesize, flags);
    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(hostPtr, nullptr);

    // Clean up
    result = hipHostFree(hostPtr);
    ASSERT_EQ(result, hipSuccess);
}

// Test hipHostGetDevicePointer
TEST_F(CudaApiTest, CuMemHostGetDevicePointer) {
    hipError_t result;
    hipDeviceptr_t devPtr;
    void *hostPtr;
    unsigned int flags = 0;

    // First allocate host memory
    result = hipHostAlloc(&hostPtr, 1024, flags);
    ASSERT_EQ(result, hipSuccess);

    result = hipHostGetDevicePointer(&devPtr, hostPtr, flags);
    ASSERT_EQ(result, hipSuccess);
    ASSERT_NE(devPtr, 0);

    // Clean up
    result = hipHostFree(hostPtr);
    ASSERT_EQ(result, hipSuccess);
}

// Test hipMemMap
TEST_F(CudaApiTest, CuMemMap) {
    hipError_t result;
    hipDeviceptr_t ptr = 0;
    unsigned int flags = 0;
    size_t offset = 0;

    // 查询设备属性
    size_t granularity = 0;
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;

    hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);

    // 确保大小是粒度的整数倍
    size_t size = ((1024 * 1024) + granularity - 1) & ~(granularity - 1); // 1MB对齐

    // 创建内存分配
    hipMemGenericAllocationHandle_t handle;
    result = hipMemCreate(&handle, size, &prop, 0);
    ASSERT_EQ(result, hipSuccess);

    // Reserve an address
    result = hipMemAddressReserve(&ptr, size, 4096, 0, flags);
    ASSERT_EQ(result, hipSuccess);

    result = hipMemMap(ptr, size, offset, handle, flags);
    ASSERT_EQ(result, hipSuccess);

    // Clean up
    result = hipMemUnmap(ptr, size);
    ASSERT_EQ(result, hipSuccess);

    result = hipMemAddressFree(ptr, size);
    ASSERT_EQ(result, hipSuccess);

    result = hipMemRelease(handle);
    ASSERT_EQ(result, hipSuccess);
}

// Test hipMemPoolImportPointer
TEST_F(CudaApiTest, CuMemPoolImportPointer) {
    hipError_t result;
    hipDeviceptr_t ptr_out = 0;
    hipMemPool_t pool = nullptr;
    hipMemPoolPtrExportData shareData = {};

    result = hipMemPoolImportPointer(&ptr_out, pool, &shareData);
    // This test is expected to fail since we don't have a valid memory pool
    ASSERT_NE(result, hipSuccess);
}

// Test hipMemRelease
TEST_F(CudaApiTest, CuMemRelease) {
    hipError_t result;

    // 查询设备属性
    size_t granularity = 0;
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;

    hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);

    // 确保大小是粒度的整数倍
    size_t size = ((1024 * 1024) + granularity - 1) & ~(granularity - 1); // 1MB对齐

    // 创建内存分配
    hipMemGenericAllocationHandle_t handle;
    result = hipMemCreate(&handle, size, &prop, 0);
    ASSERT_EQ(result, hipSuccess);

    result = hipMemRelease(handle);
    ASSERT_EQ(result, hipSuccess);
}

// Test hipModuleGetGlobal
TEST_F(CudaApiTest, CuModuleGetGlobal) {
    hipError_t result;
    hipDeviceptr_t devPtr = 0;
    size_t bytes = 0;
    hipModule_t hmod = nullptr;
    const char *name = "test";

    result = hipModuleGetGlobal(&devPtr, &bytes, hmod, name);
    // This test is expected to fail since we don't have a valid module handle
    ASSERT_NE(result, hipSuccess);
}

// Test hipDrvPointerGetAttributes
TEST_F(CudaApiTest, CuPointerGetAttributes) {
    hipError_t result;

    // 分配设备内存
    size_t size = 1024 * 1024; // 1MB
    hipDeviceptr_t d_ptr;
    hipMalloc(&d_ptr, size);

    // 准备查询指针属性
    hipPointer_attribute attributes[3] = {HIP_POINTER_ATTRIBUTE_CONTEXT, HIP_POINTER_ATTRIBUTE_MEMORY_TYPE, HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL};

    // 查询指针属性
    void *data[3];
    hipCtx_t ctx;
    unsigned int mem_type;
    int device_ordinal;

    data[0] = &ctx;
    data[1] = &mem_type;
    data[2] = &device_ordinal;

    result = hipDrvPointerGetAttributes(3,          // 属性数量
                                    attributes, // 属性数组
                                    data,       // 结果数据数组
                                    d_ptr       // 要查询的指针
    );
    ASSERT_EQ(result, hipSuccess);

    // Clean up
    result = hipFree(d_ptr);
    ASSERT_EQ(result, hipSuccess);
}

// Test hipTexRefGetAddress
TEST_F(CudaApiTest, CuTexRefGetAddress) {
    hipError_t result;
    hipDeviceptr_t devPtr = 0;
    hipTexRef hTexRef = nullptr;

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations"
    result = hipTexRefGetAddress(&devPtr, hTexRef);
#pragma GCC diagnostic pop

    // This test is expected to fail since we don't have a valid texture reference
    ASSERT_NE(result, hipSuccess);
}

// Test hipGraphMemFreeNodeGetParams
TEST_F(CudaApiTest, CuGraphMemFreeNodeGetParams) {
    hipError_t result;

    // Test 1: Invalid node handle
    {
        hipGraphNode_t hNode = nullptr;
        hipDeviceptr_t dptr_out = 0;
        result = hipGraphMemFreeNodeGetParams(hNode, &dptr_out);
        ASSERT_NE(result, hipSuccess);
    }

    // Test 2: Create a valid graph and memory free node
    {
        // Create a graph
        hipGraph_t graph;
        result = hipGraphCreate(&graph, 0);
        ASSERT_EQ(result, hipSuccess) << "Failed to create graph";

        // Allocate device memory
        hipDeviceptr_t dptr;
        result = hipMalloc(&dptr, 1024);
        ASSERT_EQ(result, hipSuccess) << "Failed to allocate device memory";

        // Create a memory free node
        hipGraphNode_t hNode;
        hipGraphNode_t dependencies[] = {}; // Empty dependencies array
        result = hipDrvGraphAddMemFreeNode(&hNode, graph, dependencies, 0, dptr);
        if(result != hipSuccess) {
            const char *errorName;
            hipDrvGetErrorName(result, &errorName);
            const char *errorString;
            hipDrvGetErrorString(result, &errorString);
            FAIL() << "Failed to create memory free node: " << errorName << " - " << errorString;
        }

        // Get node parameters
        hipDeviceptr_t dptr_out = 0;
        result = hipGraphMemFreeNodeGetParams(hNode, &dptr_out);
        ASSERT_EQ(result, hipSuccess) << "Failed to get node parameters";
        ASSERT_EQ(dptr_out, dptr) << "Device pointer mismatch";

        // Clean up
        result = hipGraphDestroy(graph);
        ASSERT_EQ(result, hipSuccess) << "Failed to destroy graph";
    }

    // Test 3: Create multiple memory free nodes in a graph
    {
        // Create a graph
        hipGraph_t graph;
        result = hipGraphCreate(&graph, 0);
        ASSERT_EQ(result, hipSuccess) << "Failed to create graph";

        // Allocate multiple device memory blocks
        const int numBlocks = 3;
        hipDeviceptr_t dptrs[numBlocks];
        hipGraphNode_t nodes[numBlocks];
        hipGraphNode_t dependencies[] = {}; // Empty dependencies array

        for(int i = 0; i < numBlocks; i++) {
            // Allocate device memory
            result = hipMalloc(&dptrs[i], 1024);
            ASSERT_EQ(result, hipSuccess) << "Failed to allocate device memory block " << i;

            // Create a memory free node
            result = hipDrvGraphAddMemFreeNode(&nodes[i], graph, dependencies, 0, dptrs[i]);
            if(result != hipSuccess) {
                const char *errorName;
                hipDrvGetErrorName(result, &errorName);
                const char *errorString;
                hipDrvGetErrorString(result, &errorString);
                FAIL() << "Failed to create memory free node " << i << ": " << errorName << " - " << errorString;
            }

            // Get node parameters
            hipDeviceptr_t dptr_out = 0;
            result = hipGraphMemFreeNodeGetParams(nodes[i], &dptr_out);
            ASSERT_EQ(result, hipSuccess) << "Failed to get node parameters for node " << i;
            ASSERT_EQ(dptr_out, dptrs[i]) << "Device pointer mismatch for node " << i;
        }

        // Clean up
        result = hipGraphDestroy(graph);
        ASSERT_EQ(result, hipSuccess) << "Failed to destroy graph";

        // Free device memory
        for(int i = 0; i < numBlocks; i++) {
            result = hipFree(dptrs[i]);
            ASSERT_EQ(result, hipSuccess) << "Failed to free device memory block " << i;
        }
    }

    // Test 4: Create a graph with dependencies between memory free nodes
    {
        // Create a graph
        hipGraph_t graph;
        result = hipGraphCreate(&graph, 0);
        ASSERT_EQ(result, hipSuccess) << "Failed to create graph";

        // Allocate device memory
        hipDeviceptr_t dptr1, dptr2;
        result = hipMalloc(&dptr1, 1024);
        ASSERT_EQ(result, hipSuccess) << "Failed to allocate first device memory block";
        result = hipMalloc(&dptr2, 1024);
        ASSERT_EQ(result, hipSuccess) << "Failed to allocate second device memory block";

        // Create memory free nodes
        hipGraphNode_t hNode1, hNode2;
        hipGraphNode_t dependencies1[] = {}; // Empty dependencies array for first node
        result = hipDrvGraphAddMemFreeNode(&hNode1, graph, dependencies1, 0, dptr1);
        if(result != hipSuccess) {
            const char *errorName;
            hipDrvGetErrorName(result, &errorName);
            const char *errorString;
            hipDrvGetErrorString(result, &errorString);
            FAIL() << "Failed to create first memory free node: " << errorName << " - " << errorString;
        }

        hipGraphNode_t dependencies2[] = {hNode1}; // Second node depends on first node
        result = hipDrvGraphAddMemFreeNode(&hNode2, graph, dependencies2, 1, dptr2);
        if(result != hipSuccess) {
            const char *errorName;
            hipDrvGetErrorName(result, &errorName);
            const char *errorString;
            hipDrvGetErrorString(result, &errorString);
            FAIL() << "Failed to create second memory free node: " << errorName << " - " << errorString;
        }

        // Get node parameters
        hipDeviceptr_t dptr_out1 = 0, dptr_out2 = 0;
        result = hipGraphMemFreeNodeGetParams(hNode1, &dptr_out1);
        ASSERT_EQ(result, hipSuccess) << "Failed to get first node parameters";
        ASSERT_EQ(dptr_out1, dptr1) << "First device pointer mismatch";

        result = hipGraphMemFreeNodeGetParams(hNode2, &dptr_out2);
        ASSERT_EQ(result, hipSuccess) << "Failed to get second node parameters";
        ASSERT_EQ(dptr_out2, dptr2) << "Second device pointer mismatch";

        // Clean up
        result = hipGraphDestroy(graph);
        ASSERT_EQ(result, hipSuccess) << "Failed to destroy graph";

        // Free device memory
        result = hipFree(dptr1);
        ASSERT_EQ(result, hipSuccess) << "Failed to free first device memory block";
        result = hipFree(dptr2);
        ASSERT_EQ(result, hipSuccess) << "Failed to free second device memory block";
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
