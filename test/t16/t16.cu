#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Simple vector addition kernel
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

void checkCudaError(hipError_t err, const char *msg) {
    if(err != hipSuccess) {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Vector size
    const int N = 1024;
    size_t size = N * sizeof(float);

    // Host arrays
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize input arrays
    for(int i = 0; i < N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Device arrays
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc(&d_A, size), "hipMalloc d_A failed");
    checkCudaError(hipMalloc(&d_B, size), "hipMalloc d_B failed");
    checkCudaError(hipMalloc(&d_C, size), "hipMalloc d_C failed");

    // Check compute capability
    int device;
    hipGetDevice(&device);
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
    int computeCapability = major * 10 + minor;
    if(computeCapability < 35) {
        fprintf(stderr, "Device compute capability %d.%d is less than 3.5\n", major, minor);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        exit(EXIT_FAILURE);
    }
    printf("Device compute capability: %d.%d\n", major, minor);

    // Create CUDA graph
    hipGraph_t graph;
    checkCudaError(hipGraphCreate(&graph, 0), "hipGraphCreate failed");

    // Graph nodes
    hipGraphNode_t memcpyNodeA, memcpyNodeB, kernelNode;

    // Add memcpy node for A (host to device)
    hipMemcpy3DParms memcpyParamsA = {0};
    memcpyParamsA.srcPtr = make_hipPitchedPtr((void *)h_A, size, N, 1);
    memcpyParamsA.dstPtr = make_hipPitchedPtr((void *)d_A, size, N, 1);
    memcpyParamsA.extent = make_hipExtent(size, 1, 1);
    memcpyParamsA.kind = hipMemcpyHostToDevice;
    checkCudaError(hipGraphAddMemcpyNode(&memcpyNodeA, graph, NULL, 0, &memcpyParamsA), "hipGraphAddMemcpyNode A failed");

    // Add memcpy node for B (host to device)
    hipMemcpy3DParms memcpyParamsB = {0};
    memcpyParamsB.srcPtr = make_hipPitchedPtr((void *)h_B, size, N, 1);
    memcpyParamsB.dstPtr = make_hipPitchedPtr((void *)d_B, size, N, 1);
    memcpyParamsB.extent = make_hipExtent(size, 1, 1);
    memcpyParamsB.kind = hipMemcpyHostToDevice;
    checkCudaError(hipGraphAddMemcpyNode(&memcpyNodeB, graph, NULL, 0, &memcpyParamsB), "hipGraphAddMemcpyNode B failed");

    // Add kernel node
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    void *kernelArgs[] = {(void *)&d_A, (void *)&d_B, (void *)&d_C, (void *)&N};
    hipKernelNodeParams kernelParams = {0};
    kernelParams.func = (void *)vectorAdd;
    kernelParams.gridDim = grid;
    kernelParams.blockDim = block;
    kernelParams.sharedMemBytes = 0;
    kernelParams.kernelParams = kernelArgs;
    kernelParams.extra = NULL;
    checkCudaError(hipGraphAddKernelNode(&kernelNode, graph, NULL, 0, &kernelParams), "hipGraphAddKernelNode failed");

    // Add dependencies: kernelNode depends on memcpyNodeA and memcpyNodeB
    checkCudaError(hipGraphAddDependencies(graph, &memcpyNodeA, &kernelNode, 1), "hipGraphAddDependencies A failed");
    checkCudaError(hipGraphAddDependencies(graph, &memcpyNodeB, &kernelNode, 1), "hipGraphAddDependencies B failed");

    // Instantiate and launch the graph
    hipGraphExec_t graphExec;
    checkCudaError(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0), "hipGraphInstantiate failed");
    checkCudaError(hipGraphLaunch(graphExec, 0), "hipGraphLaunch failed");

    // Copy result back to host
    checkCudaError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost), "hipMemcpy failed");

    // Verify result
    for(int i = 0; i < N; i++) {
        if(fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Verification failed at index %d\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Vector addition completed successfully!\n");

    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
