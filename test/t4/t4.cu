#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(call)                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                 \
    do {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                       \
        hipError_t err = call;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                   \
        if(err != hipSuccess) {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                              \
            const char *errStr;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                \
            hipDrvGetErrorString(err, &errStr);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                    \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, errStr);                                                                                                                                                                                                                                                                                                                                                                                                                                         \
            exit(EXIT_FAILURE);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                \
        }                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                      \
    } while(0)

int main() {
    // 初始化CUDA驱动API
    CHECK_CUDA_ERROR(hipInit(0));

    // 获取设备
    hipDevice_t device;
    CHECK_CUDA_ERROR(hipDeviceGet(&device, 0));

    // 创建上下文
    hipCtx_t context;
    CHECK_CUDA_ERROR(hipCtxCreate(&context, 0, device));

    // 查询设备属性
    size_t granularity = 0;
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = device;

    CHECK_CUDA_ERROR(hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum));

    printf("Minimum allocation granularity: %zu bytes\n", granularity);

    // 确保大小是粒度的整数倍
    size_t size = ((1024 * 1024) + granularity - 1) & ~(granularity - 1); // 1MB对齐

    // 创建内存分配
    hipMemGenericAllocationHandle_t handle;
    CHECK_CUDA_ERROR(hipMemCreate(&handle, size, &prop, 0));

    printf("Successfully created memory allocation of size %zu bytes\n", size);

    // 获取分配的内存物理地址
    hipDeviceptr_t ptr;
    CHECK_CUDA_ERROR(hipMemAddressReserve(&ptr, size, 0, 0, 0));
    CHECK_CUDA_ERROR(hipMemMap(ptr, size, 0, handle, 0));

    // 设置访问权限
    hipMemAccessDesc accessDesc = {};
    accessDesc.location.type = hipMemLocationTypeDevice;
    accessDesc.location.id = device;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    CHECK_CUDA_ERROR(hipMemSetAccess(ptr, size, &accessDesc, 1));

    // 使用分配的内存...
    printf("Memory successfully allocated and mapped at address: %p\n", (void *)ptr);

    // 清理资源
    CHECK_CUDA_ERROR(hipMemUnmap(ptr, size));
    CHECK_CUDA_ERROR(hipMemAddressFree(ptr, size));
    CHECK_CUDA_ERROR(hipMemRelease(handle));
    CHECK_CUDA_ERROR(hipCtxDestroy(context));

    return 0;
}
