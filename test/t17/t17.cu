#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Global device symbol
__device__ float d_symbol[1024];

void checkCudaError(hipError_t err, const char *msg) {
    if(err != hipSuccess) {
        fprintf(stderr, "%s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Data size
    const int N = 1024;
    size_t size = N * sizeof(float);

    // Host arrays
    float *h_input = (float *)malloc(size);
    float *h_output = (float *)malloc(size);

    // Initialize input array
    for(int i = 0; i < N; i++) {
        h_input[i] = (float)i;
        h_output[i] = 0.0f;
    }

    // Device buffer
    float *d_buffer;
    checkCudaError(hipMalloc(&d_buffer, size), "hipMalloc d_buffer failed");

    // Check compute capability
    int device;
    hipGetDevice(&device);
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
    int computeCapability = major * 10 + minor;
    if(computeCapability < 35) {
        fprintf(stderr, "Device compute capability %d.%d is less than 3.5\n", major, minor);
        hipFree(d_buffer);
        free(h_input);
        free(h_output);
        exit(EXIT_FAILURE);
    }
    printf("Device compute capability: %d.%d\n", major, minor);

    // Get symbol address
    void *symbol_addr;
    checkCudaError(hipGetSymbolAddress(&symbol_addr, HIP_SYMBOL(d_symbol)), "hipGetSymbolAddress failed");

    // Create CUDA graph
    hipGraph_t graph;
    checkCudaError(hipGraphCreate(&graph, 0), "hipGraphCreate failed");

    // Graph nodes
    hipGraphNode_t memcpyToSymbolNode, memcpyFromSymbolNode;

    // Add memcpy node to copy from host to device symbol
    checkCudaError(hipGraphAddMemcpyNodeToSymbol(&memcpyToSymbolNode, graph, NULL, 0, HIP_SYMBOL(d_symbol), h_input, size, 0, hipMemcpyHostToDevice), "hipGraphAddMemcpyNodeToSymbol failed");

    // Add memcpy node to copy from device symbol to device buffer
    checkCudaError(hipGraphAddMemcpyNodeFromSymbol(&memcpyFromSymbolNode, graph, &memcpyToSymbolNode, 1, d_buffer, HIP_SYMBOL(d_symbol), size, 0, hipMemcpyDeviceToDevice), "hipGraphAddMemcpyNodeFromSymbol failed");

    // Instantiate and launch the graph
    hipGraphExec_t graphExec;
    checkCudaError(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0), "hipGraphInstantiate failed");
    checkCudaError(hipGraphLaunch(graphExec, 0), "hipGraphLaunch failed");

    // Copy result from device buffer to host
    checkCudaError(hipMemcpy(h_output, d_buffer, size, hipMemcpyDeviceToHost), "hipMemcpy failed");

    // Verify result
    for(int i = 0; i < N; i++) {
        if(fabs(h_input[i] - h_output[i]) > 1e-5) {
            fprintf(stderr, "Verification failed at index %d: expected %f, got %f\n", i, h_input[i], h_output[i]);
            hipGraphExecDestroy(graphExec);
            hipGraphDestroy(graph);
            hipFree(d_buffer);
            free(h_input);
            free(h_output);
            exit(EXIT_FAILURE);
        }
    }
    printf("Symbol copy completed successfully!\n");

    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_buffer);
    free(h_input);
    free(h_output);

    return 0;
}
