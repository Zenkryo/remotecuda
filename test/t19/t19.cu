#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ float d_symbol[1024];

int main() {
    const int N = 1024;
    size_t size = N * sizeof(float);

    float *h_input = (float *)malloc(size);
    float *h_output = (float *)malloc(size);

    for(int i = 0; i < N; i++) {
        h_input[i] = (float)i;
        h_output[i] = 0.0f;
    }

    float *d_buffer;
    hipMalloc(&d_buffer, size);

    // int device;
    // cudaGetDevice(&device);
    // int major, minor;
    // cudaDeviceGetAttribute(&major, cudaDevAttrComputeCapabilityMajor, device);
    // cudaDeviceGetAttribute(&minor, cudaDevAttrComputeCapabilityMinor, device);
    // int computeCapability = major * 10 + minor;
    // if(computeCapability < 35) {
    //     fprintf(stderr, "Device compute capability %d.%d is less than 3.5\n", major, minor);
    //     cudaFree(d_buffer);
    //     free(h_input);
    //     free(h_output);
    //     exit(EXIT_FAILURE);
    // }
    void *symbol_addr;
    hipGetSymbolAddress(&symbol_addr, HIP_SYMBOL(d_symbol));

    hipGraph_t graph;
    hipGraphCreate(&graph, 0);

    hipGraphNode_t memcpyToSymbolNode, memcpyFromSymbolNode;

    // Add memcpy node to copy from host to device symbol
    hipGraphAddMemcpyNodeToSymbol(&memcpyToSymbolNode, graph, NULL, 0, HIP_SYMBOL(d_symbol), h_input, size, 0, hipMemcpyHostToDevice);

    // Add memcpy node to copy from device symbol to device buffer
    hipGraphAddMemcpyNodeFromSymbol(&memcpyFromSymbolNode, graph, &memcpyToSymbolNode, 1, d_buffer, HIP_SYMBOL(d_symbol), size, 0, hipMemcpyDeviceToDevice);

    // Instantiate and launch the graph
    hipGraphExec_t graphExec;
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    hipGraphLaunch(graphExec, 0);

    // Copy result from device buffer to host
    hipMemcpy(h_output, d_buffer, size, hipMemcpyDeviceToHost);

    // Verify result

    for(int i = 0; i < N; i++) {
        if(fabs(h_input[i] - h_output[i]) > 1e-5) {
            fprintf(stderr, "Verification failed at index %d: expected %f, got %f\n", i, h_input[i], h_output[i]);
            hipGraphExecDestroy(graphExec);
            hipGraphDestroy(graph);
            hipFree(d_buffer);
            free(h_input);
            free(h_output);
            exit(EXIT_FAILURE);
        }
    }
    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_buffer);
    free(h_input);
    free(h_output);
}
