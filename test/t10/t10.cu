#include <hip/hip_runtime.h>
#include <stdio.h>

// Simple CUDA kernel
__global__ void addKernel(int *c, const int *a, const int *b) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    c[idx] = a[idx] + b[idx];
}

int main() {
    // Error checking macro
    #define CUDA_CHECK(call) { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1); \
        } \
    }

    // Initialize data
    const int N = 1024;
    size_t size = N * sizeof(int);
    
    int *h_a = (int*)malloc(size);
    int *h_b = (int*)malloc(size);
    int *h_c = (int*)malloc(size);
    
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device memory
    int *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(&d_a, size));
    CUDA_CHECK(hipMalloc(&d_b, size));
    CUDA_CHECK(hipMalloc(&d_c, size));

    CUDA_CHECK(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    // Create CUDA graph
    hipGraph_t graph;
    CUDA_CHECK(hipGraphCreate(&graph, 0));

    // Create kernel node parameters
    hipKernelNodeParams kernelNodeParams = {0};
    void* kernelArgs[] = { &d_c, &d_a, &d_b };
    
    kernelNodeParams.func = (void*)addKernel;
    kernelNodeParams.gridDim = dim3(N/256, 1, 1);
    kernelNodeParams.blockDim = dim3(256, 1, 1);
    kernelNodeParams.sharedMemBytes = 0;
    kernelNodeParams.kernelParams = kernelArgs;
    kernelNodeParams.extra = NULL;

    // Add kernel node to graph
    hipGraphNode_t kernelNode;
    CUDA_CHECK(hipGraphAddKernelNode(&kernelNode, graph, NULL, 0, &kernelNodeParams));

    // Create child graph
    hipGraph_t childGraph;
    CUDA_CHECK(hipGraphCreate(&childGraph, 0));

    // Add child graph node
    hipGraphNode_t childGraphNode;
    CUDA_CHECK(hipGraphAddChildGraphNode(&childGraphNode, graph, &kernelNode, 1, childGraph));

    // Instantiate and launch graph
    hipGraphExec_t graphExec;
    CUDA_CHECK(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
    CUDA_CHECK(hipGraphLaunch(graphExec, 0));
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back
    CUDA_CHECK(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

    // Verify result
    for (int i = 0; i < N; i++) {
        if (h_c[i] != h_a[i] + h_b[i]) {
            printf("Verification failed at index %d: %d != %d + %d\n", 
                   i, h_c[i], h_a[i], h_b[i]);
            break;
        }
    }
    printf("Verification completed successfully\n");

    // Cleanup
    CUDA_CHECK(hipGraphExecDestroy(graphExec));
    CUDA_CHECK(hipGraphDestroy(childGraph));
    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
