#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphHostNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipGraphNode_t hostNode;
    hipHostNodeParams hostParams = {0};
    hostParams.fn = [](void *userData) {};
    hostParams.userData = nullptr;
    err = hipGraphAddHostNode(&hostNode, graph, nullptr, 0, &hostParams);
    CHECK_CUDA_ERROR(err, "Failed to add host node");
    hipHostNodeParams retrievedParams;
    err = hipGraphHostNodeGetParams(hostNode, &retrievedParams);
    CHECK_CUDA_ERROR(err, "Failed to get host node parameters");
    ASSERT_EQ(retrievedParams.fn, hostParams.fn) << "Function pointer mismatch";
    hostParams.fn = [](void *userData) {};
    err = hipGraphHostNodeSetParams(hostNode, &hostParams);
    CHECK_CUDA_ERROR(err, "Failed to set host node parameters");
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
}
