#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphCreate){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    ASSERT_NE(graph, nullptr);
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
}
