#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGetChannelDesc){
    const int width = 32;
    const int height = 32;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array;
    hipError_t err = hipMallocArray(&array, &channelDesc, width, height);
    CHECK_CUDA_ERROR(err, "Failed to allocate CUDA array");
    hipChannelFormatDesc retrievedDesc;
    err = hipGetChannelDesc(&retrievedDesc, array);
    CHECK_CUDA_ERROR(err, "Failed to get channel description");
    ASSERT_EQ(retrievedDesc.x, channelDesc.x) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.y, channelDesc.y) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.z, channelDesc.z) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.w, channelDesc.w) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.f, channelDesc.f) << "Channel format mismatch";
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free CUDA array");
}
