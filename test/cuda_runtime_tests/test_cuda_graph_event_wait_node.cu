#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphEventWaitNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipEvent_t event;
    err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    hipGraphNode_t eventWaitNode;
    err = hipGraphAddEventWaitNode(&eventWaitNode, graph, nullptr, 0, event);
    CHECK_CUDA_ERROR(err, "Failed to add event wait node");
    hipEvent_t retrievedEvent;
    err = hipGraphEventWaitNodeGetEvent(eventWaitNode, &retrievedEvent);
    CHECK_CUDA_ERROR(err, "Failed to get event");
    ASSERT_EQ(retrievedEvent, event) << "Event mismatch";
    hipEvent_t newEvent;
    err = hipEventCreate(&newEvent);
    CHECK_CUDA_ERROR(err, "Failed to create new event");
    err = hipGraphEventWaitNodeSetEvent(eventWaitNode, newEvent);
    CHECK_CUDA_ERROR(err, "Failed to set event wait node event");
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
    err = hipEventDestroy(newEvent);
    CHECK_CUDA_ERROR(err, "Failed to destroy new event");
}
