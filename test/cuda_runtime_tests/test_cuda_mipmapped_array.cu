#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMipmappedArray){
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipExtent extent = make_hipExtent(32, 32, 0);
    unsigned int numLevels = 5;
    hipMipmappedArray_t mipmappedArray;
    hipError_t err = hipMallocMipmappedArray(&mipmappedArray, &channelDesc, extent, numLevels);
    CHECK_CUDA_ERROR(err, "Failed to allocate mipmapped array");
    ASSERT_NE(mipmappedArray, nullptr);
    for(unsigned int level = 0; level < numLevels; level++) {
        hipArray_t levelArray;
        err = hipGetMipmappedArrayLevel(&levelArray, mipmappedArray, level);
        CHECK_CUDA_ERROR(err, (std::string("Failed to get mipmap level ") + std::to_string(level)).c_str());
        ASSERT_NE(levelArray, nullptr);
    }
    err = hipFreeMipmappedArray(mipmappedArray);
    CHECK_CUDA_ERROR(err, "Failed to free mipmapped array");
}
