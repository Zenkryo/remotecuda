#include "hip/hip_runtime.h"
#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphOperations) {
    hipError_t err;
    const int N = 1024;
    size_t size = N * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    for(int i = 0; i < N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    float *d_A, *d_B, *d_C;
    err = hipMalloc(&d_A, size);
    CHECK_CUDA_ERROR(err, "hipMalloc d_A failed");
    err = hipMalloc(&d_B, size);
    CHECK_CUDA_ERROR(err, "hipMalloc d_B failed");
    err = hipMalloc(&d_C, size);
    CHECK_CUDA_ERROR(err, "hipMalloc d_C failed");
    int device;
    hipGetDevice(&device);
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
    int computeCapability = major * 10 + minor;
    if(computeCapability < 35) {
        fprintf(stderr, "Device compute capability %d.%d is less than 3.5\n", major, minor);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C);
        exit(EXIT_FAILURE);
    }
    hipGraph_t graph;
    err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "hipGraphCreate failed");
    hipGraphNode_t memcpyNodeA, memcpyNodeB, kernelNode;
    hipMemcpy3DParms memcpyParamsA = {0};
    memcpyParamsA.srcPtr = make_hipPitchedPtr((void *)h_A, size, N, 1);
    memcpyParamsA.dstPtr = make_hipPitchedPtr((void *)d_A, size, N, 1);
    memcpyParamsA.extent = make_hipExtent(size, 1, 1);
    memcpyParamsA.kind = hipMemcpyHostToDevice;
    err = hipGraphAddMemcpyNode(&memcpyNodeA, graph, NULL, 0, &memcpyParamsA);
    CHECK_CUDA_ERROR(err, "hipGraphAddMemcpyNode A failed");
    hipMemcpy3DParms memcpyParamsB = {0};
    memcpyParamsB.srcPtr = make_hipPitchedPtr((void *)h_B, size, N, 1);
    memcpyParamsB.dstPtr = make_hipPitchedPtr((void *)d_B, size, N, 1);
    memcpyParamsB.extent = make_hipExtent(size, 1, 1);
    memcpyParamsB.kind = hipMemcpyHostToDevice;
    err = hipGraphAddMemcpyNode(&memcpyNodeB, graph, NULL, 0, &memcpyParamsB);
    CHECK_CUDA_ERROR(err, "hipGraphAddMemcpyNode B failed");
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    void *kernelArgs[] = {(void *)&d_A, (void *)&d_B, (void *)&d_C, (void *)&N};
    hipKernelNodeParams kernelParams = {0};
    kernelParams.func = (void *)vectorAdd;
    kernelParams.gridDim = grid;
    kernelParams.blockDim = block;
    kernelParams.sharedMemBytes = 0;
    kernelParams.kernelParams = kernelArgs;
    kernelParams.extra = NULL;
    err = hipGraphAddKernelNode(&kernelNode, graph, NULL, 0, &kernelParams);
    CHECK_CUDA_ERROR(err, "hipGraphAddKernelNode failed");
    err = hipGraphAddDependencies(graph, &memcpyNodeA, &kernelNode, 1);
    CHECK_CUDA_ERROR(err, "hipGraphAddDependencies A failed");
    err = hipGraphAddDependencies(graph, &memcpyNodeB, &kernelNode, 1);
    CHECK_CUDA_ERROR(err, "hipGraphAddDependencies B failed");
    hipGraphExec_t graphExec;
    err = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    CHECK_CUDA_ERROR(err, "hipGraphInstantiate failed");
    err = hipGraphLaunch(graphExec, 0);
    CHECK_CUDA_ERROR(err, "hipGraphLaunch failed");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err, "hipMemcpy failed");
    for(int i = 0; i < N; i++) {
        if(fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Verification failed at index %d\n", i);
            exit(EXIT_FAILURE);
        }
    }
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}
