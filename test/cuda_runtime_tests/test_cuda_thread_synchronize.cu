#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaThreadSynchronize){
    hipError_t err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err, "Failed to synchronize thread");
}
