#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamAttachMemAsync){
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    void *ptr;
    err = hipMallocManaged(&ptr, 1024, hipMemAttachGlobal);
    if(err == hipSuccess) {
        err = hipStreamAttachMemAsync(stream, ptr, 0, hipMemAttachGlobal);
        CHECK_CUDA_ERROR(err, "Failed to attach memory to stream");
        err = hipFree(ptr);
        CHECK_CUDA_ERROR(err, "Failed to free managed memory");
    } else {
        SUCCEED() << "Managed memory not supported, skipping test";
    }
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
