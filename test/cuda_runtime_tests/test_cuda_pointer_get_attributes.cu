#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaPointerGetAttributes){
    const int size = 1024;
    void *devPtr;
    hipError_t err = hipMalloc(&devPtr, size);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    hipPointerAttribute_t attr;
    err = hipPointerGetAttributes(&attr, devPtr);
    CHECK_CUDA_ERROR(err, "Failed to get pointer attributes");
    ASSERT_EQ(attr.type, hipMemoryTypeDevice) << "Invalid memory type";
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
}
