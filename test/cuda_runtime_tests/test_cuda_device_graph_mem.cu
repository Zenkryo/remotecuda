#include "hip/hip_runtime.h"
#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceGraphMem){
    hipError_t err = hipDeviceGraphMemTrim(0);
    CHECK_CUDA_ERROR(err, "Failed to trim graph memory");
    int value;
    err = hipDeviceGetAttribute(&value, hipDeviceAttributeMaxThreadsPerBlock, 0);
    if(err == hipSuccess) {
        ASSERT_GT(value, 0) << "Invalid max threads per block";
    }
}
