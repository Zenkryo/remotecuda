#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpy3DPeer){
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    CHECK_CUDA_ERROR(err, "Failed to get device count");
    if(deviceCount > 1) {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        hipExtent extent = make_hipExtent(32, 32, 32);
        hipArray_t srcArray, dstArray;
        err = hipSetDevice(0);
        CHECK_CUDA_ERROR(err, "Failed to set source device");
        err = hipMalloc3DArray(&srcArray, &channelDesc, extent);
        CHECK_CUDA_ERROR(err, "Failed to allocate source array");
        err = hipSetDevice(1);
        CHECK_CUDA_ERROR(err, "Failed to set destination device");
        err = hipMalloc3DArray(&dstArray, &channelDesc, extent);
        CHECK_CUDA_ERROR(err, "Failed to allocate destination array");
        err = hipDeviceEnablePeerAccess(0, 0);
        if(err == hipSuccess) {
            hipMemcpy3DPeerParms copyParams = {0};
            copyParams.srcArray = srcArray;
            copyParams.srcDevice = 0;
            copyParams.dstArray = dstArray;
            copyParams.dstDevice = 1;
            copyParams.extent = extent;
            err = hipMemcpy3DPeer(&copyParams);
            CHECK_CUDA_ERROR(err, "Failed to perform 3D peer memory copy");
            err = hipDeviceDisablePeerAccess(0);
            CHECK_CUDA_ERROR(err, "Failed to disable peer access");
        }
        err = hipSetDevice(0);
        CHECK_CUDA_ERROR(err, "Failed to set device 0");
        err = hipFreeArray(srcArray);
        CHECK_CUDA_ERROR(err, "Failed to free source array");
        err = hipSetDevice(1);
        CHECK_CUDA_ERROR(err, "Failed to set device 1");
        err = hipFreeArray(dstArray);
        CHECK_CUDA_ERROR(err, "Failed to free destination array");
    } else {
        SUCCEED() << "Skipping test - requires multiple devices";
    }
}
