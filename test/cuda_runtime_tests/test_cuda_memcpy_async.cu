#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpyAsync){
    const int size = 1024;
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    float *hostSrc = new float[size];
    float *hostDst = new float[size];
    float *devPtr;
    err = hipMalloc(&devPtr, size * sizeof(float));
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    for(int i = 0; i < size; i++) {
        hostSrc[i] = static_cast<float>(i);
    }
    err = hipMemcpyAsync(devPtr, hostSrc, size * sizeof(float), hipMemcpyHostToDevice, stream);
    CHECK_CUDA_ERROR(err, "Failed to copy from host to device");
    err = hipMemcpyAsync(hostDst, devPtr, size * sizeof(float), hipMemcpyDeviceToHost, stream);
    CHECK_CUDA_ERROR(err, "Failed to copy from device to host");
    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stream");
    for(int i = 0; i < size; i++) {
        ASSERT_EQ(hostDst[i], hostSrc[i]) << "Memory copy failed at index " << i;
    }
    delete[] hostSrc;
    delete[] hostDst;
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
