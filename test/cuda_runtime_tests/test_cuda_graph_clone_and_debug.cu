#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphCloneAndDebug){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipGraphNode_t node;
    void *kernelArgs[] = {NULL};
    hipKernelNodeParams kernelParams = {0};
    kernelParams.func = (void *)test_kernel;
    kernelParams.gridDim = dim3(1, 1, 1);
    kernelParams.blockDim = dim3(1, 1, 1);
    kernelParams.sharedMemBytes = 0;
    kernelParams.kernelParams = kernelArgs;
    kernelParams.extra = NULL;
    err = hipGraphAddKernelNode(&node, graph, NULL, 0, &kernelParams);
    CHECK_CUDA_ERROR(err, "Failed to add kernel node");
    hipGraph_t clonedGraph;
    err = hipGraphClone(&clonedGraph, graph);
    CHECK_CUDA_ERROR(err, "Failed to clone graph");
    err = hipGraphDebugDotPrint(graph, "original_graph.dot", 0);
    if(err != hipErrorNotSupported) {
        CHECK_CUDA_ERROR(err, "Failed to print debug dot file");
    }
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy original graph");
    err = hipGraphDestroy(clonedGraph);
    CHECK_CUDA_ERROR(err, "Failed to destroy cloned graph");
}
