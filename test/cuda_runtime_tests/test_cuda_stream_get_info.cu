#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamGetInfo){
    hipStream_t stream;
    hipError_t err;
    int leastPriority, greatestPriority;
    err = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
    CHECK_CUDA_ERROR(err, "Failed to get stream priority range");
    err = hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, greatestPriority);
    CHECK_CUDA_ERROR(err, "Failed to create stream with priority");
    int priority;
    err = hipStreamGetPriority(stream, &priority);
    CHECK_CUDA_ERROR(err, "Failed to get stream priority");
    ASSERT_EQ(priority, greatestPriority) << "Stream priority not set correctly";
    unsigned int flags;
    err = hipStreamGetFlags(stream, &flags);
    CHECK_CUDA_ERROR(err, "Failed to get stream flags");
    ASSERT_EQ(flags, hipStreamNonBlocking) << "Stream flags not set correctly";
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
