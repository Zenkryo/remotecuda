#include "common.h"

TEST_F(CudaRuntimeApiTest, Cuda3DArray){
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipExtent extent = make_hipExtent(32, 32, 32);
    hipArray_t array;
    hipError_t err = hipMalloc3DArray(&array, &channelDesc, extent);
    CHECK_CUDA_ERROR(err, "Failed to allocate 3D array");
    ASSERT_NE(array, nullptr);
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free 3D array");
}
