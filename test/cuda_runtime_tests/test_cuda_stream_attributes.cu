#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamAttributes){
    hipStream_t srcStream, dstStream;
    hipError_t err;
    err = hipStreamCreate(&srcStream);
    CHECK_CUDA_ERROR(err, "Failed to create source stream");
    err = hipStreamCreate(&dstStream);
    CHECK_CUDA_ERROR(err, "Failed to create destination stream");
    hipLaunchAttributeValue value;
    value.accessPolicyWindow.base_ptr = nullptr;
    value.accessPolicyWindow.num_bytes = 0;
    value.accessPolicyWindow.hitRatio = 1.0f;
    value.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
    value.accessPolicyWindow.missProp = hipAccessPropertyStreaming;
    err = hipStreamSetAttribute(srcStream, hipLaunchAttributeAccessPolicyWindow, &value);
    if(err == hipSuccess) {
        err = cudaStreamCopyAttributes(dstStream, srcStream);
        CHECK_CUDA_ERROR(err, "Failed to copy stream attributes");
        hipLaunchAttributeValue retrievedValue;
        err = hipStreamGetAttribute(dstStream, hipLaunchAttributeAccessPolicyWindow, &retrievedValue);
        CHECK_CUDA_ERROR(err, "Failed to get stream attribute");
        ASSERT_EQ(retrievedValue.accessPolicyWindow.hitRatio, value.accessPolicyWindow.hitRatio) << "Stream attribute not copied correctly";
    } else {
        SUCCEED() << "Stream attributes not supported, skipping test";
    }
    err = hipStreamDestroy(srcStream);
    CHECK_CUDA_ERROR(err, "Failed to destroy source stream");
    err = hipStreamDestroy(dstStream);
    CHECK_CUDA_ERROR(err, "Failed to destroy destination stream");
}
