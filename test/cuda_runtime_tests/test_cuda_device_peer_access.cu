#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDevicePeerAccess){
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    CHECK_CUDA_ERROR(err, "Failed to get device count");
    if(deviceCount > 1) {
        int canAccessPeer;
        err = hipDeviceCanAccessPeer(&canAccessPeer, 0, 1);
        CHECK_CUDA_ERROR(err, "Failed to check peer access");
        if(canAccessPeer) {
            err = hipDeviceEnablePeerAccess(1, 0);
            if(err == hipSuccess) {
                err = hipDeviceDisablePeerAccess(1);
                CHECK_CUDA_ERROR(err, "Failed to disable peer access");
            }
        }
    }
}
