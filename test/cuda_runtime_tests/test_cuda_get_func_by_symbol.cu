#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGetFuncBySymbol){
    hipFunction_t func;
    hipError_t err = hipGetFuncBySymbol(&func, (const void *)test_kernel);
    if(err == hipSuccess) {
        ASSERT_NE(func, nullptr) << "Invalid function pointer";
    } else {
        SUCCEED() << "Function symbol lookup not supported, skipping test";
    }
}
