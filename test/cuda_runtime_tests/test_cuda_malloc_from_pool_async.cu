#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMallocFromPoolAsync){
    hipMemPool_t memPool = nullptr;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0; // Current device
    poolProps.location.type = hipMemLocationTypeDevice;
    hipError_t err = hipMemPoolCreate(&memPool, &poolProps);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pools not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to create memory pool");
    hipStream_t stream;
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    void *devPtr;
    err = hipMallocFromPoolAsync(&devPtr, 1024, memPool, stream);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pool allocation not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to allocate memory from pool");
    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stream");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
    err = hipMemPoolDestroy(memPool);
    CHECK_CUDA_ERROR(err, "Failed to destroy memory pool");
}
