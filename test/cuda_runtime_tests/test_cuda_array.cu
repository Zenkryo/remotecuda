#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaArray){
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array;
    hipError_t err = hipMallocArray(&array, &channelDesc, 32, 32);
    CHECK_CUDA_ERROR(err, "Failed to allocate array");
    ASSERT_NE(array, nullptr);
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free array");
}
