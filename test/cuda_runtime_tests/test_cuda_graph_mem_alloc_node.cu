#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphMemAllocNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipGraphNode_t allocNode;
    hipMemAllocNodeParams allocParams = {};
    allocParams.poolProps.allocType = hipMemAllocationTypePinned;
    allocParams.poolProps.location.type = hipMemLocationTypeDevice;
    allocParams.poolProps.location.id = 0;
    allocParams.bytesize = 1024;
    allocParams.dptr = nullptr;
    err = hipGraphAddMemAllocNode(&allocNode, graph, nullptr, 0, &allocParams);
    if(err == hipSuccess) {
        hipMemAllocNodeParams retrievedParams;
        err = hipGraphMemAllocNodeGetParams(allocNode, &retrievedParams);
        CHECK_CUDA_ERROR(err, "Failed to get memory allocation node parameters");
        ASSERT_EQ(retrievedParams.bytesize, allocParams.bytesize) << "Allocation size mismatch";
    }
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
}
