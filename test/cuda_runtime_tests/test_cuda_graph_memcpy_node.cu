#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphMemcpyNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    void *hostPtr = malloc(1024);
    ASSERT_NE(hostPtr, nullptr);
    void *devPtr;
    err = hipMalloc(&devPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    hipGraphNode_t memcpyNode;
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr(hostPtr, 1024, 1024, 1);
    copyParams.dstPtr = make_hipPitchedPtr(devPtr, 1024, 1024, 1);
    copyParams.extent = make_hipExtent(1024, 1, 1);
    copyParams.kind = hipMemcpyHostToDevice;
    err = hipGraphAddMemcpyNode(&memcpyNode, graph, nullptr, 0, &copyParams);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Graph memcpy node not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to add memcpy node");
    hipMemcpy3DParms retrievedParams;
    err = hipGraphMemcpyNodeGetParams(memcpyNode, &retrievedParams);
    CHECK_CUDA_ERROR(err, "Failed to get memcpy node parameters");
    ASSERT_EQ(retrievedParams.srcPtr.ptr, copyParams.srcPtr.ptr) << "Source pointer mismatch";
    ASSERT_EQ(retrievedParams.dstPtr.ptr, copyParams.dstPtr.ptr) << "Destination pointer mismatch";
    copyParams.srcPtr = make_hipPitchedPtr(devPtr, 1024, 1024, 1);
    copyParams.dstPtr = make_hipPitchedPtr(hostPtr, 1024, 1024, 1);
    copyParams.extent = make_hipExtent(1024, 1, 1);
    copyParams.kind = hipMemcpyDeviceToHost;
    err = hipGraphMemcpyNodeSetParams(memcpyNode, &copyParams);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Graph memcpy node parameter setting not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to set memcpy node parameters");
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
    free(hostPtr);
}
