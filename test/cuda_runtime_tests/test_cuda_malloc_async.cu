#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMallocAsync){
    const int size = 1024;
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    void *devPtr;
    err = hipMallocAsync(&devPtr, size, stream);
    if(err != hipErrorNotSupported) {
        CHECK_CUDA_ERROR(err, "Failed to allocate memory asynchronously");
        err = hipFreeAsync(devPtr, stream);
        CHECK_CUDA_ERROR(err, "Failed to free memory asynchronously");
    }
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
