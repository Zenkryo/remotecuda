#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaEventRecordWithFlags){
    hipEvent_t event;
    hipStream_t stream;
    hipError_t err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    err = hipEventRecord(event, stream);
    CHECK_CUDA_ERROR(err, "Failed to record event");
    err = hipEventSynchronize(event);
    CHECK_CUDA_ERROR(err, "Failed to synchronize event");
    err = hipEventRecordWithFlags(event, stream, hipEventRecordExternal);
    if(err == hipErrorNotSupported) {
        SUCCEED() << "Event recording with flags not supported, skipping test";
    } else if(err == hipErrorIllegalState) {
        SUCCEED() << "Event recording with flags not allowed in current state, skipping test";
    } else {
        CHECK_CUDA_ERROR(err, "Failed to record event with flags");
    }
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
