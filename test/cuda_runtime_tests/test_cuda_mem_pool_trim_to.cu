#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemPoolTrimTo){
    hipMemPool_t memPool = nullptr;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0; // Current device
    poolProps.location.type = hipMemLocationTypeDevice;
    hipError_t err = hipMemPoolCreate(&memPool, &poolProps);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pools not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to create memory pool");
    err = hipMemPoolTrimTo(memPool, 1024 * 1024);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pool trim not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to trim memory pool");
    err = hipMemPoolDestroy(memPool);
    CHECK_CUDA_ERROR(err, "Failed to destroy memory pool");
}
