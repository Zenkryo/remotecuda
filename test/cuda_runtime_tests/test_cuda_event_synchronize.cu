#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaEventSynchronize){
    hipEvent_t start, stop;
    hipStream_t stream;
    hipError_t err;
    err = hipEventCreate(&start);
    CHECK_CUDA_ERROR(err, "Failed to create start event");
    err = hipEventCreate(&stop);
    CHECK_CUDA_ERROR(err, "Failed to create stop event");
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    err = hipEventRecord(start, stream);
    CHECK_CUDA_ERROR(err, "Failed to record start event");
    test_kernel<<<1, 1, 0, stream>>>();
    err = hipEventRecord(stop, stream);
    CHECK_CUDA_ERROR(err, "Failed to record stop event");
    err = hipEventSynchronize(stop);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stop event");
    float elapsedTime;
    err = hipEventElapsedTime(&elapsedTime, start, stop);
    CHECK_CUDA_ERROR(err, "Failed to get elapsed time");
    ASSERT_GT(elapsedTime, 0.0f) << "Invalid elapsed time";
    err = hipEventDestroy(start);
    CHECK_CUDA_ERROR(err, "Failed to destroy start event");
    err = hipEventDestroy(stop);
    CHECK_CUDA_ERROR(err, "Failed to destroy stop event");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
