#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphMemsetNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    void *devPtr;
    err = hipMalloc(&devPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    hipGraphNode_t memsetNode;
    hipMemsetParams memsetParams = {0};
    memsetParams.dst = devPtr;
    memsetParams.elementSize = 1;
    memsetParams.width = 1024;
    memsetParams.height = 1;
    memsetParams.value = 0x42;
    err = hipGraphAddMemsetNode(&memsetNode, graph, nullptr, 0, &memsetParams);
    CHECK_CUDA_ERROR(err, "Failed to add memset node");
    hipMemsetParams retrievedParams;
    err = hipGraphMemsetNodeGetParams(memsetNode, &retrievedParams);
    CHECK_CUDA_ERROR(err, "Failed to get memset node parameters");
    ASSERT_EQ(retrievedParams.dst, memsetParams.dst) << "Destination pointer mismatch";
    ASSERT_EQ(retrievedParams.value, memsetParams.value) << "Value mismatch";
    memsetParams.value = 0x84;
    err = hipGraphMemsetNodeSetParams(memsetNode, &memsetParams);
    CHECK_CUDA_ERROR(err, "Failed to set memset node parameters");
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
}
