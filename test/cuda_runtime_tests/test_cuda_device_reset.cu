#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceReset){
    hipError_t err = hipDeviceReset();
    CHECK_CUDA_ERROR(err, "Failed to reset device");
}
