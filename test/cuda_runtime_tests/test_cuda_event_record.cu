#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaEventRecord){
    hipEvent_t event;
    hipStream_t stream;
    hipError_t err;
    err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    test_kernel<<<1, 1, 0, stream>>>();
    err = hipEventRecord(event, stream);
    CHECK_CUDA_ERROR(err, "Failed to record event");
    err = hipEventQuery(event);
    if(err == hipSuccess) {
        SUCCEED() << "Event completed faster than expected, but this is acceptable";
    } else {
        ASSERT_EQ(err, hipErrorNotReady) << "Unexpected error from hipEventQuery";
    }
    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stream");
    err = hipEventQuery(event);
    ASSERT_EQ(err, hipSuccess) << "Event should be complete after stream synchronization";
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
