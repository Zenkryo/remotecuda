#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpyArray){
    const int size = 1024;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array;
    hipError_t err = hipMallocArray(&array, &channelDesc, size);
    CHECK_CUDA_ERROR(err, "Failed to allocate CUDA array");
    float *hostSrc = new float[size];
    float *hostDst = new float[size];
    for(int i = 0; i < size; i++) {
        hostSrc[i] = static_cast<float>(i);
    }
    err = hipMemcpyToArray(array, 0, 0, hostSrc, size * sizeof(float), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err, "Failed to copy from host to array");
    err = hipMemcpyFromArray(hostDst, array, 0, 0, size * sizeof(float), hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err, "Failed to copy from array to host");
    for(int i = 0; i < size; i++) {
        ASSERT_EQ(hostDst[i], hostSrc[i]) << "Memory copy failed at index " << i;
    }
    delete[] hostSrc;
    delete[] hostDst;
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free CUDA array");
}
