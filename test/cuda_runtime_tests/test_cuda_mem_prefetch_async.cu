#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemPrefetchAsync){
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    void *devPtr;
    err = hipMalloc(&devPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    err = hipMemPrefetchAsync(devPtr, 1024, 0, stream);
    if(err != hipSuccess) {
        SUCCEED() << "Memory prefetch not supported on this device, skipping test";
    }
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
}
