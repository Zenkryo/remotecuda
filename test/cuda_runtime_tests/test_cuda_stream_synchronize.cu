#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamSynchronize){
    hipStream_t stream;
    hipError_t err;
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    test_kernel<<<1, 1, 0, stream>>>();
    err = hipStreamQuery(stream);
    if(err == hipSuccess) {
        SUCCEED() << "Stream completed faster than expected, but this is acceptable";
    } else {
        ASSERT_EQ(err, hipErrorNotReady) << "Unexpected error from hipStreamQuery";
    }
    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stream");
    err = hipStreamQuery(stream);
    ASSERT_EQ(err, hipSuccess) << "Stream should be complete after synchronization";
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
