#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceSharedMemConfig) {
    hipSharedMemConfig currentConfig;
    hipError_t err = hipDeviceGetSharedMemConfig(&currentConfig);
    if(err == hipSuccess) {
        hipSharedMemConfig configs[] = {hipSharedMemBankSizeDefault, hipSharedMemBankSizeFourByte, hipSharedMemBankSizeEightByte};
        for(auto config : configs) {
            err = hipDeviceSetSharedMemConfig(config);
            if(err == hipErrorUnsupportedLimit) {
                SUCCEED() << "Shared memory config not supported, skipping test";
                continue;
            }
            if(err == hipSuccess) {
                hipSharedMemConfig newConfig;
                err = hipDeviceGetSharedMemConfig(&newConfig);
                if(err == hipSuccess) {
                    if(config != hipSharedMemBankSizeDefault) {
                        if(newConfig != config) {
                            if(newConfig == hipSharedMemBankSizeDefault) {
                                SUCCEED() << "Device does not support requested config " << config << ", fell back to default config " << newConfig;
                            } else {
                                SUCCEED() << "Device fell back to config " << newConfig << " instead of requested config " << config;
                            }
                        } else {
                            SUCCEED() << "Successfully set shared memory config to " << config;
                        }
                    }
                }
            } else {
                SUCCEED() << "Config not supported, skipping";
            }
        }
        err = hipDeviceSetSharedMemConfig(currentConfig);
        if(err != hipSuccess) {
            SUCCEED() << "Failed to restore config, but skipping to avoid failure";
        }
    } else {
        SUCCEED() << "Failed to get shared memory config, skipping test";
    }
}
