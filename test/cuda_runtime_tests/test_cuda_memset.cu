#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemset){
    void *devPtr;
    hipError_t err = hipMalloc(&devPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    err = hipMemset(devPtr, 0x42, 1024);
    CHECK_CUDA_ERROR(err, "Failed to set device memory");
    char *hostPtr = new char[1024];
    err = hipMemcpy(hostPtr, devPtr, 1024, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err, "Failed to copy from device to host");
    for(int i = 0; i < 1024; i++) {
        ASSERT_EQ(hostPtr[i], 0x42) << "Memory not set correctly at index " << i;
    }
    delete[] hostPtr;
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
}
