#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpyArrayAsync){
    const int size = 1024;
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array;
    err = hipMallocArray(&array, &channelDesc, size);
    CHECK_CUDA_ERROR(err, "Failed to allocate CUDA array");
    float *hostSrc = new float[size];
    float *hostDst = new float[size];
    for(int i = 0; i < size; i++) {
        hostSrc[i] = static_cast<float>(i);
    }
    err = cudaMemcpyToArrayAsync(array, 0, 0, hostSrc, size * sizeof(float), hipMemcpyHostToDevice, stream);
    CHECK_CUDA_ERROR(err, "Failed to copy from host to array");
    err = cudaMemcpyFromArrayAsync(hostDst, array, 0, 0, size * sizeof(float), hipMemcpyDeviceToHost, stream);
    CHECK_CUDA_ERROR(err, "Failed to copy from array to host");
    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stream");
    for(int i = 0; i < size; i++) {
        ASSERT_EQ(hostDst[i], hostSrc[i]) << "Memory copy failed at index " << i;
    }
    delete[] hostSrc;
    delete[] hostDst;
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free CUDA array");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
