#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphKernelNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipGraphNode_t kernelNode;
    void *kernelArgs[] = {};
    hipKernelNodeParams nodeParams = {};
    nodeParams.func = (void *)test_kernel;
    nodeParams.gridDim = dim3(1, 1, 1);
    nodeParams.blockDim = dim3(1, 1, 1);
    nodeParams.sharedMemBytes = 0;
    nodeParams.kernelParams = kernelArgs;
    nodeParams.extra = nullptr;
    err = hipGraphAddKernelNode(&kernelNode, graph, nullptr, 0, &nodeParams);
    CHECK_CUDA_ERROR(err, "Failed to add kernel node");
    hipKernelNodeParams retrievedParams;
    err = hipGraphKernelNodeGetParams(kernelNode, &retrievedParams);
    CHECK_CUDA_ERROR(err, "Failed to get kernel node parameters");
    ASSERT_EQ(retrievedParams.func, nodeParams.func) << "Kernel function mismatch";
    ASSERT_EQ(retrievedParams.gridDim.x, nodeParams.gridDim.x) << "Grid dimension mismatch";
    ASSERT_EQ(retrievedParams.blockDim.x, nodeParams.blockDim.x) << "Block dimension mismatch";
    nodeParams.gridDim = dim3(2, 2, 1);
    err = hipGraphKernelNodeSetParams(kernelNode, &nodeParams);
    CHECK_CUDA_ERROR(err, "Failed to set kernel node parameters");
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
}
