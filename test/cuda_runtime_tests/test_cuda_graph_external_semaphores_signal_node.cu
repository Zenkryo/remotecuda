#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphExternalSemaphoresSignalNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipGraphNode_t signalNode;
    hipExternalSemaphoreSignalNodeParams signalParams = {0};
    signalParams.extSemArray = nullptr;
    signalParams.paramsArray = nullptr;
    signalParams.numExtSems = 0;
    err = hipGraphAddExternalSemaphoresSignalNode(&signalNode, graph, nullptr, 0, &signalParams);
    if(err == hipSuccess) {
        hipExternalSemaphoreSignalNodeParams retrievedParams;
        err = hipGraphExternalSemaphoresSignalNodeGetParams(signalNode, &retrievedParams);
        CHECK_CUDA_ERROR(err, "Failed to get external semaphores signal node parameters");
        err = hipGraphExternalSemaphoresSignalNodeSetParams(signalNode, &signalParams);
        CHECK_CUDA_ERROR(err, "Failed to set external semaphores signal node parameters");
    }
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
}
