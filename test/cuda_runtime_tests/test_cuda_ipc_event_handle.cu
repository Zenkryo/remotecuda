#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaIpcEventHandle){
    hipEvent_t event;
    hipError_t err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    hipIpcEventHandle_t handle;
    err = hipIpcGetEventHandle(&handle, event);
    if(err == hipSuccess) { // Only proceed if IPC is supported
        hipEvent_t openedEvent;
        err = hipIpcOpenEventHandle(&openedEvent, handle);
        CHECK_CUDA_ERROR(err, "Failed to open IPC event handle");
        err = hipEventDestroy(openedEvent);
        CHECK_CUDA_ERROR(err, "Failed to destroy opened event");
    }
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
}
