#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaArrayGetInfo){
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array;
    hipError_t err = hipMallocArray(&array, &channelDesc, 32, 32);
    CHECK_CUDA_ERROR(err, "Failed to allocate array");
    hipChannelFormatDesc retrievedDesc;
    hipExtent extent;
    unsigned int flags;
    err = hipArrayGetInfo(&retrievedDesc, &extent, &flags, array);
    CHECK_CUDA_ERROR(err, "Failed to get array info");
    ASSERT_EQ(retrievedDesc.x, channelDesc.x) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.y, channelDesc.y) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.z, channelDesc.z) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.w, channelDesc.w) << "Channel format mismatch";
    ASSERT_EQ(retrievedDesc.f, channelDesc.f) << "Channel format mismatch";
    ASSERT_EQ(extent.width, 32) << "Width mismatch";
    ASSERT_EQ(extent.height, 32) << "Height mismatch";
    ASSERT_EQ(extent.depth, 0) << "Depth mismatch";
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free array");
}
