#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphExternalSemaphoresWaitNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipGraphNode_t waitNode;
    hipExternalSemaphoreWaitNodeParams waitParams = {0};
    waitParams.extSemArray = nullptr;
    waitParams.paramsArray = nullptr;
    waitParams.numExtSems = 0;
    err = hipGraphAddExternalSemaphoresWaitNode(&waitNode, graph, nullptr, 0, &waitParams);
    if(err == hipSuccess) {
        hipExternalSemaphoreWaitNodeParams retrievedParams;
        err = hipGraphExternalSemaphoresWaitNodeGetParams(waitNode, &retrievedParams);
        CHECK_CUDA_ERROR(err, "Failed to get external semaphores wait node parameters");
        err = hipGraphExternalSemaphoresWaitNodeSetParams(waitNode, &waitParams);
        CHECK_CUDA_ERROR(err, "Failed to set external semaphores wait node parameters");
    }
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
}
