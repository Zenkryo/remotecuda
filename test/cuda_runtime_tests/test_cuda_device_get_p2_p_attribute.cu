#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceGetP2PAttribute){
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    CHECK_CUDA_ERROR(err, "Failed to get device count");
    if(deviceCount > 1) {
        int value;
        err = hipDeviceGetP2PAttribute(&value, hipDevP2PAttrPerformanceRank, 0, 1);
        if(err == hipSuccess) {
            SUCCEED() << "P2P attributes retrieved successfully";
        } else if(err == hipErrorInvalidDevice) {
            SUCCEED() << "P2P not supported between these devices";
        } else {
            CHECK_CUDA_ERROR(err, "Failed to get P2P attribute");
        }
    } else {
        SUCCEED() << "Skipping test - requires multiple devices";
    }
}
