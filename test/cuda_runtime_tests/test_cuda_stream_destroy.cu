#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamDestroy){
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
