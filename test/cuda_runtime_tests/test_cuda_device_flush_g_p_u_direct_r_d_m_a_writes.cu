#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceFlushGPUDirectRDMAWrites){
    hipError_t err = cudaDeviceFlushGPUDirectRDMAWrites(cudaFlushGPUDirectRDMAWritesTargetCurrentDevice, cudaFlushGPUDirectRDMAWritesToOwner);
    if(err != hipErrorNotSupported) {
        CHECK_CUDA_ERROR(err, "Failed to flush GPU Direct RDMA writes");
    }
}
