#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpy3DAsync){
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipExtent extent = make_hipExtent(32, 32, 32);
    hipArray_t srcArray, dstArray;
    hipError_t err = hipMalloc3DArray(&srcArray, &channelDesc, extent);
    CHECK_CUDA_ERROR(err, "Failed to allocate source array");
    err = hipMalloc3DArray(&dstArray, &channelDesc, extent);
    CHECK_CUDA_ERROR(err, "Failed to allocate destination array");
    hipStream_t stream;
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcArray = srcArray;
    copyParams.dstArray = dstArray;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    err = hipMemcpy3DAsync(&copyParams, stream);
    CHECK_CUDA_ERROR(err, "Failed to perform async 3D memory copy");
    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stream");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
    err = hipFreeArray(srcArray);
    CHECK_CUDA_ERROR(err, "Failed to free source array");
    err = hipFreeArray(dstArray);
    CHECK_CUDA_ERROR(err, "Failed to free destination array");
}
