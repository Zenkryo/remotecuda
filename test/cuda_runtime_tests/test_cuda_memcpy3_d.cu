#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpy3D){
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipExtent extent = make_hipExtent(32, 32, 32);
    hipArray_t srcArray, dstArray;
    hipError_t err = hipMalloc3DArray(&srcArray, &channelDesc, extent);
    CHECK_CUDA_ERROR(err, "Failed to allocate source array");
    err = hipMalloc3DArray(&dstArray, &channelDesc, extent);
    CHECK_CUDA_ERROR(err, "Failed to allocate destination array");
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcArray = srcArray;
    copyParams.dstArray = dstArray;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    err = hipMemcpy3D(&copyParams);
    CHECK_CUDA_ERROR(err, "Failed to perform 3D memory copy");
    err = hipFreeArray(srcArray);
    CHECK_CUDA_ERROR(err, "Failed to free source array");
    err = hipFreeArray(dstArray);
    CHECK_CUDA_ERROR(err, "Failed to free destination array");
}
