#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpy2DArrayToArray){
    const int width = 32;
    const int height = 32;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t srcArray, dstArray;
    hipError_t err = hipMallocArray(&srcArray, &channelDesc, width, height);
    CHECK_CUDA_ERROR(err, "Failed to allocate source array");
    err = hipMallocArray(&dstArray, &channelDesc, width, height);
    CHECK_CUDA_ERROR(err, "Failed to allocate destination array");
    float *hostData = new float[width * height];
    for(int i = 0; i < width * height; i++) {
        hostData[i] = static_cast<float>(i);
    }
    err = hipMemcpy2DToArray(srcArray, 0, 0, hostData, width * sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err, "Failed to copy from host to source array");
    err = hipMemcpy2DArrayToArray(dstArray, 0, 0, srcArray, 0, 0, width * sizeof(float), height, hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(err, "Failed to copy from source array to destination array");
    delete[] hostData;
    err = hipFreeArray(srcArray);
    CHECK_CUDA_ERROR(err, "Failed to free source array");
    err = hipFreeArray(dstArray);
    CHECK_CUDA_ERROR(err, "Failed to free destination array");
}
