#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphEmptyNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipGraphNode_t emptyNode;
    err = hipGraphAddEmptyNode(&emptyNode, graph, nullptr, 0);
    CHECK_CUDA_ERROR(err, "Failed to add empty node");
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
}
