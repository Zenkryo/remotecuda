#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphEventRecordNode){
    hipGraph_t graph;
    hipError_t err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create graph");
    hipEvent_t event;
    err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    hipGraphNode_t eventRecordNode;
    err = hipGraphAddEventRecordNode(&eventRecordNode, graph, nullptr, 0, event);
    CHECK_CUDA_ERROR(err, "Failed to add event record node");
    hipEvent_t retrievedEvent;
    err = hipGraphEventRecordNodeGetEvent(eventRecordNode, &retrievedEvent);
    CHECK_CUDA_ERROR(err, "Failed to get event");
    ASSERT_EQ(retrievedEvent, event) << "Event mismatch";
    hipEvent_t newEvent;
    err = hipEventCreate(&newEvent);
    CHECK_CUDA_ERROR(err, "Failed to create new event");
    err = hipGraphEventRecordNodeSetEvent(eventRecordNode, newEvent);
    CHECK_CUDA_ERROR(err, "Failed to set event record node event");
    err = hipGraphDestroy(graph);
    CHECK_CUDA_ERROR(err, "Failed to destroy graph");
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
    err = hipEventDestroy(newEvent);
    CHECK_CUDA_ERROR(err, "Failed to destroy new event");
}
