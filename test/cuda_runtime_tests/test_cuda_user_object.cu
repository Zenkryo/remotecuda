#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaUserObject){
    hipUserObject_t userObject;
    int data = 42;
    hipError_t err = hipUserObjectCreate(
        &userObject, &data,
        [](void *ptr) {
            int *data = static_cast<int *>(ptr);
            *data = 0;
        },
        1, hipUserObjectNoDestructorSync);
    if(err == hipSuccess) {
        err = hipUserObjectRetain(userObject);
        CHECK_CUDA_ERROR(err, "Failed to retain user object");
        hipGraph_t graph;
        err = hipGraphCreate(&graph, 0);
        CHECK_CUDA_ERROR(err, "Failed to create graph");
        err = hipGraphRetainUserObject(graph, userObject);
        CHECK_CUDA_ERROR(err, "Failed to retain user object in graph");
        err = hipGraphReleaseUserObject(graph, userObject);
        CHECK_CUDA_ERROR(err, "Failed to release user object from graph");
        err = hipUserObjectRelease(userObject);
        CHECK_CUDA_ERROR(err, "Failed to release user object");
        err = hipGraphDestroy(graph);
        CHECK_CUDA_ERROR(err, "Failed to destroy graph");
    } else {
        SUCCEED() << "User objects not supported, skipping test";
    }
}
