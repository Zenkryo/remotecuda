#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamCreate){
    hipStream_t stream;
    hipError_t err;
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
    err = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    CHECK_CUDA_ERROR(err, "Failed to create stream with flags");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
    int leastPriority, greatestPriority;
    err = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
    CHECK_CUDA_ERROR(err, "Failed to get stream priority range");
    err = hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, greatestPriority);
    CHECK_CUDA_ERROR(err, "Failed to create stream with priority");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
