#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceLimits){
    struct TestCase {
        hipLimit_t limit;
        size_t value;
        const char *description;
    };
    TestCase testCases[] = {{hipLimitStackSize, 4096, "Stack size"}, {hipLimitPrintfFifoSize, 1048576, "Printf FIFO size"}, {hipLimitMallocHeapSize, 8 * 1024 * 1024, "Malloc heap size"}};
    for(const auto &testCase : testCases) {
        size_t currentValue;
        hipError_t err = hipDeviceGetLimit(&currentValue, testCase.limit);
        CHECK_CUDA_ERROR(err, (std::string("Failed to get ") + testCase.description + " limit").c_str());
        err = hipDeviceSetLimit(testCase.limit, testCase.value);
        CHECK_CUDA_ERROR(err, (std::string("Failed to set ") + testCase.description + " limit").c_str());
        size_t newValue;
        err = hipDeviceGetLimit(&newValue, testCase.limit);
        CHECK_CUDA_ERROR(err, (std::string("Failed to verify ") + testCase.description + " limit").c_str());
        ASSERT_GE(newValue, testCase.value) << "Failed to set " << testCase.description << " limit";
        err = hipDeviceSetLimit(testCase.limit, currentValue);
        CHECK_CUDA_ERROR(err, (std::string("Failed to restore ") + testCase.description + " limit").c_str());
    }
}
