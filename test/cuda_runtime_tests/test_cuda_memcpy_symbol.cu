#include "common.h"

__device__ float g_dev_symbol;

TEST_F(CudaRuntimeApiTest, CudaMemcpySymbol) {
    float hostValue = 42.0f;
    float retrievedValue = 0.0f;
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(g_dev_symbol), &hostValue, sizeof(float));
    CHECK_CUDA_ERROR(err, "Failed to copy to symbol");
    err = hipMemcpyFromSymbol(&retrievedValue, HIP_SYMBOL(g_dev_symbol), sizeof(float));
    CHECK_CUDA_ERROR(err, "Failed to copy from symbol");
    ASSERT_EQ(retrievedValue, hostValue) << "Symbol copy failed";
}
