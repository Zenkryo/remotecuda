#include "hip/hip_runtime.h"
#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceGraphMemAttributes){
    hipError_t err;
    int deviceCount;
    err = hipGetDeviceCount(&deviceCount);
    CHECK_CUDA_ERROR(err, "Failed to get device count");
    for(int device = 0; device < deviceCount; ++device) {
        hipSetDevice(device);
        int computeCapabilityMajor;
        int computeCapabilityMinor;
        int maxThreadsPerBlock;
        int sharedMemPerBlock;
        int maxThreadsPerMultiProcessor;
        int multiProcessorCount;
        int maxGridSize[3];
        int maxThreadsDim[3];
        int warpSize;
        err = hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, device);
        CHECK_CUDA_ERROR(err, "Failed to get compute capability major");
        err = hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, device);
        CHECK_CUDA_ERROR(err, "Failed to get compute capability minor");
        err = hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device);
        CHECK_CUDA_ERROR(err, "Failed to get max threads per block");
        err = hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, device);
        CHECK_CUDA_ERROR(err, "Failed to get warp size");
        err = hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, device);
        CHECK_CUDA_ERROR(err, "Failed to get multi processor count");
        err = hipDeviceGetAttribute(&maxThreadsPerMultiProcessor, hipDeviceAttributeMaxThreadsPerMultiProcessor, device);
        CHECK_CUDA_ERROR(err, "Failed to get max threads per multi processor");
        err = hipDeviceGetAttribute(&maxGridSize[0], hipDeviceAttributeMaxGridDimX, device);
        CHECK_CUDA_ERROR(err, "Failed to get max grid size x");
        err = hipDeviceGetAttribute(&maxGridSize[2], hipDeviceAttributeMaxGridDimZ, device);
        CHECK_CUDA_ERROR(err, "Failed to get max grid size z");
        err = hipDeviceGetAttribute(&maxThreadsDim[0], hipDeviceAttributeMaxBlockDimX, device);
        CHECK_CUDA_ERROR(err, "Failed to get max block dim x");
        err = hipDeviceGetAttribute(&maxThreadsDim[1], hipDeviceAttributeMaxBlockDimY, device);
        CHECK_CUDA_ERROR(err, "Failed to get max block dim y");
        err = hipDeviceGetAttribute(&maxThreadsDim[2], hipDeviceAttributeMaxBlockDimZ, device);
        CHECK_CUDA_ERROR(err, "Failed to get max block dim z");
        err = hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
        CHECK_CUDA_ERROR(err, "Failed to get shared memory per block");
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, device);
        CHECK_CUDA_ERROR(err, "Failed to get device properties");
        hipMemPool_t memPool;
        hipError_t poolErr = hipDeviceGetDefaultMemPool(&memPool, device);
        if(poolErr != hipSuccess) {
            GTEST_SKIP() << "Default memory pool not supported on this device";
        }
        uint64_t poolLowWatermark = 1024 * 1024 * 512; // 512MB
        hipError_t attrErr = hipDeviceSetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &poolLowWatermark);
        if(attrErr == hipSuccess) {
            uint64_t retrievedLowWatermark;
            err = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &retrievedLowWatermark);
            CHECK_CUDA_ERROR(err, "Failed to get graph memory attribute");
            uint64_t usedMem;
            uint64_t reservedMem;
            err = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &usedMem);
            CHECK_CUDA_ERROR(err, "Failed to get graph memory attribute");
            err = hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &reservedMem);
            CHECK_CUDA_ERROR(err, "Failed to get graph memory attribute");
        } else {
            GTEST_SKIP() << "Graph memory attributes not supported on this device";
        }
    }
    err = hipDeviceReset();
    CHECK_CUDA_ERROR(err, "Failed to reset device");
}
