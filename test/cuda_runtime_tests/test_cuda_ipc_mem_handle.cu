#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaIpcMemHandle){
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if(err == hipSuccess && deviceCount > 0) {
        int device;
        err = hipGetDevice(&device);
        if(err == hipSuccess) {
            void *devPtr;
            err = hipMalloc(&devPtr, 1024);
            if(err == hipSuccess) {
                hipIpcMemHandle_t handle;
                err = hipIpcGetMemHandle(&handle, devPtr);
                if(err == hipSuccess) {
                    void *openedDevPtr;
                    err = hipIpcOpenMemHandle(&openedDevPtr, handle, hipIpcMemLazyEnablePeerAccess);
                    if(err == hipSuccess) {
                        err = hipIpcCloseMemHandle(openedDevPtr);
                        if(err != hipSuccess) {
                            SUCCEED() << "Failed to close IPC handle, but skipping";
                        }
                    } else {
                        SUCCEED() << "IPC open not supported, skipping";
                    }
                }
                err = hipFree(devPtr);
                if(err != hipSuccess) {
                    SUCCEED() << "Failed to free memory, but skipping";
                }
            } else {
                SUCCEED() << "Memory allocation failed, skipping test";
            }
        } else {
            SUCCEED() << "Failed to get device, skipping test";
        }
    } else {
        SUCCEED() << "No devices available, skipping test";
    }
}
