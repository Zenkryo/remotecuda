#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGetLastError){
    hipGetLastError();
    hipError_t peekErr = hipPeekAtLastError();
    ASSERT_EQ(peekErr, hipSuccess) << "Unexpected error from hipPeekAtLastError";
    hipError_t getErr = hipGetLastError();
    ASSERT_EQ(getErr, hipSuccess) << "Unexpected error from hipGetLastError";
    void *devPtr = nullptr;
    hipMalloc(&devPtr, (size_t)-1); // This should generate an error
    hipError_t err = hipGetLastError();
    ASSERT_NE(err, hipSuccess) << "Expected error from invalid hipMalloc";
}
