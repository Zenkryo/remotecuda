#include "hip/hip_runtime.h"
#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaFuncAttributes){
    hipError_t err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(test_kernel), hipFuncCachePreferL1);
    if(err == hipSuccess) {
        err = hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(test_kernel), hipSharedMemBankSizeFourByte);
        if(err == hipSuccess) {
            hipFuncAttributes attr;
            err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(test_kernel));
            CHECK_CUDA_ERROR(err, "Failed to get function attributes");
            ASSERT_GT(attr.maxThreadsPerBlock, 0) << "Invalid max threads per block";
            err = hipFuncSetAttribute(reinterpret_cast<const void*>(test_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, 1024);
            if(err == hipSuccess) {
                SUCCEED() << "Function attributes set successfully";
            } else {
                SUCCEED() << "Function attribute setting not supported, skipping test";
            }
        } else {
            SUCCEED() << "Shared memory config not supported, skipping test";
        }
    } else {
        SUCCEED() << "Cache config not supported, skipping test";
    }
}
