#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphChildGraphNode){
    hipGraph_t parentGraph = nullptr;
    hipGraph_t childGraph = nullptr;
    hipGraphExec_t graphExec = nullptr;
    hipError_t err;
    err = hipGraphCreate(&parentGraph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create parent graph");
    ASSERT_NE(parentGraph, nullptr);
    hipGraphNode_t kernelNode;
    void *kernelArgs[] = {};
    hipKernelNodeParams nodeParams = {};
    nodeParams.func = (void *)test_kernel;
    nodeParams.gridDim = dim3(1, 1, 1);
    nodeParams.blockDim = dim3(1, 1, 1);
    nodeParams.sharedMemBytes = 0;
    nodeParams.kernelParams = kernelArgs;
    nodeParams.extra = nullptr;
    err = hipGraphAddKernelNode(&kernelNode, parentGraph, nullptr, 0, &nodeParams);
    CHECK_CUDA_ERROR(err, "Failed to add kernel node to parent graph");
    err = hipGraphCreate(&childGraph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create child graph");
    ASSERT_NE(childGraph, nullptr);
    hipGraphNode_t childGraphNode;
    err = hipGraphAddChildGraphNode(&childGraphNode, parentGraph, &kernelNode, 1, childGraph);
    if(err == hipErrorNotSupported) {
        hipGraphDestroy(childGraph);
        hipGraphDestroy(parentGraph);
        GTEST_SKIP() << "Child graph nodes not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to add child graph node");
    ASSERT_NE(childGraphNode, nullptr);
    hipGraph_t retrievedGraph;
    err = hipGraphChildGraphNodeGetGraph(childGraphNode, &retrievedGraph);
    CHECK_CUDA_ERROR(err, "Failed to get child graph");
    ASSERT_NE(retrievedGraph, nullptr);
    err = hipGraphInstantiate(&graphExec, parentGraph, nullptr, nullptr, 0);
    CHECK_CUDA_ERROR(err, "Failed to instantiate graph");
    err = hipGraphLaunch(graphExec, nullptr);
    CHECK_CUDA_ERROR(err, "Failed to launch graph");
    err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err, "Failed to synchronize device");
    if(graphExec != nullptr) {
        err = hipGraphExecDestroy(graphExec);
        CHECK_CUDA_ERROR(err, "Failed to destroy graph execution");
    }
    if(parentGraph != nullptr) {
        err = hipGraphDestroy(parentGraph);
        CHECK_CUDA_ERROR(err, "Failed to destroy parent graph");
    }
    if(childGraph != nullptr) {
        err = hipGraphDestroy(childGraph);
        CHECK_CUDA_ERROR(err, "Failed to destroy child graph");
    }
}
