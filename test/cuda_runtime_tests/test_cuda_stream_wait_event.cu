#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamWaitEvent){
    hipStream_t stream;
    hipEvent_t event;
    hipError_t err;
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    err = hipEventRecord(event, stream);
    CHECK_CUDA_ERROR(err, "Failed to record event");
    err = hipStreamWaitEvent(stream, event, 0);
    CHECK_CUDA_ERROR(err, "Failed to wait for event");
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
