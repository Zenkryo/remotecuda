#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaArrayGetSparseProperties){
    int supportsSparseArrays = 0;
    hipError_t err = hipDeviceGetAttribute(&supportsSparseArrays, cudaDevAttrSparseCudaArraySupported, 0);
    CHECK_CUDA_ERROR(err, "Failed to get device attribute");
    if(!supportsSparseArrays) {
        GTEST_SKIP() << "Device does not support sparse arrays";
    }
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipExtent extent = make_hipExtent(32, 32, 1);
    hipArray_t array;
    err = hipMalloc3DArray(&array, &channelDesc, extent, cudaArraySparse);
    CHECK_CUDA_ERROR(err, "Failed to allocate sparse 3D array");
    cudaArraySparseProperties sparseProps;
    err = cudaArrayGetSparseProperties(&sparseProps, array);
    CHECK_CUDA_ERROR(err, "Failed to get sparse properties");
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free array");
}
