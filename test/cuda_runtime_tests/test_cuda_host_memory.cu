#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaHostMemory){
    void *hostPtr;
    hipError_t err = hipHostMalloc(&hostPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate pinned host memory");
    void *devPtr;
    err = hipHostGetDevicePointer(&devPtr, hostPtr, 0);
    if(err == hipSuccess) {
        unsigned int flags;
        err = hipHostGetFlags(&flags, hostPtr);
        CHECK_CUDA_ERROR(err, "Failed to get host memory flags");
        ASSERT_NE(flags, 0) << "Invalid host memory flags";
    } else {
        SUCCEED() << "Host device pointer not supported, skipping test";
    }
    err = hipHostFree(hostPtr);
    CHECK_CUDA_ERROR(err, "Failed to free pinned host memory");
}
