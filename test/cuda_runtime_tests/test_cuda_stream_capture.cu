#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaStreamCapture){
    hipError_t err;
    hipStream_t stream;
    err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    err = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    CHECK_CUDA_ERROR(err, "Failed to begin stream capture");
    hipStreamCaptureStatus captureStatus;
    unsigned long long graphHandle = 0;
    err = hipStreamGetCaptureInfo(stream, &captureStatus, &graphHandle);
    CHECK_CUDA_ERROR(err, "Failed to get stream capture info");
    EXPECT_EQ(captureStatus, hipStreamCaptureStatusActive);
    hipGraph_t graph = (hipGraph_t)graphHandle;
    err = hipStreamEndCapture(stream, &graph);
    CHECK_CUDA_ERROR(err, "Failed to end stream capture");
    if(graph != nullptr) {
        err = hipGraphDestroy(graph);
        CHECK_CUDA_ERROR(err, "Failed to destroy graph");
    }
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
}
