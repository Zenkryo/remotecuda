#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemPoolAccess){
    hipMemPool_t memPool = nullptr;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0; // Current device
    poolProps.location.type = hipMemLocationTypeDevice;
    hipError_t err = hipMemPoolCreate(&memPool, &poolProps);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pools not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to create memory pool");
    int deviceCount;
    err = hipGetDeviceCount(&deviceCount);
    CHECK_CUDA_ERROR(err, "Failed to get device count");
    if(deviceCount > 1) {
        hipMemAccessDesc accessDesc = {};
        accessDesc.location.type = hipMemLocationTypeDevice;
        accessDesc.location.id = 1;
        accessDesc.flags = hipMemAccessFlagsProtReadWrite;
        err = hipMemPoolSetAccess(memPool, &accessDesc, 1);
        if(err == hipErrorNotSupported) {
            GTEST_SKIP() << "Memory pool access not supported on this device";
        }
        CHECK_CUDA_ERROR(err, "Failed to set memory pool access");
        hipMemAccessFlags accessFlags;
        err = hipMemPoolGetAccess(&accessFlags, memPool, &accessDesc.location);
        CHECK_CUDA_ERROR(err, "Failed to get memory pool access");
        ASSERT_EQ(accessFlags, hipMemAccessFlagsProtReadWrite) << "Access flags not set correctly";
    }
    if(memPool != nullptr) {
        hipMemPoolDestroy(memPool);
    }
}
