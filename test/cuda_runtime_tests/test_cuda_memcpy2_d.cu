#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemcpy2D){
    const int width = 32;
    const int height = 32;
    const int pitch = width * sizeof(float);
    float *hostSrc = new float[width * height];
    float *hostDst = new float[width * height];
    void *devSrc, *devDst;
    size_t devPitch;
    hipError_t err = hipMallocPitch(&devSrc, &devPitch, width * sizeof(float), height);
    CHECK_CUDA_ERROR(err, "Failed to allocate source device memory");
    err = hipMallocPitch(&devDst, &devPitch, width * sizeof(float), height);
    CHECK_CUDA_ERROR(err, "Failed to allocate destination device memory");
    for(int i = 0; i < width * height; i++) {
        hostSrc[i] = static_cast<float>(i);
    }
    err = hipMemcpy2D(devSrc, devPitch, hostSrc, pitch, width * sizeof(float), height, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err, "Failed to copy from host to device");
    err = hipMemcpy2D(devDst, devPitch, devSrc, devPitch, width * sizeof(float), height, hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(err, "Failed to copy from device to device");
    err = hipMemcpy2D(hostDst, pitch, devDst, devPitch, width * sizeof(float), height, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err, "Failed to copy from device to host");
    for(int i = 0; i < width * height; i++) {
        ASSERT_EQ(hostDst[i], hostSrc[i]) << "Memory copy failed at index " << i;
    }
    delete[] hostSrc;
    delete[] hostDst;
    err = hipFree(devSrc);
    CHECK_CUDA_ERROR(err, "Failed to free source device memory");
    err = hipFree(devDst);
    CHECK_CUDA_ERROR(err, "Failed to free destination device memory");
}
