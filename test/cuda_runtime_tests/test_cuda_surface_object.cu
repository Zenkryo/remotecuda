#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaSurfaceObject){
    const int width = 32;
    const int height = 32;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array;
    hipError_t err = hipMallocArray(&array, &channelDesc, width, height);
    CHECK_CUDA_ERROR(err, "Failed to allocate CUDA array");
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;
    hipSurfaceObject_t surfObj;
    err = hipCreateSurfaceObject(&surfObj, &resDesc);
    if(err == hipSuccess) {
        hipResourceDesc retrievedResDesc;
        err = cudaGetSurfaceObjectResourceDesc(&retrievedResDesc, surfObj);
        CHECK_CUDA_ERROR(err, "Failed to get surface object resource descriptor");
        err = hipDestroySurfaceObject(surfObj);
        CHECK_CUDA_ERROR(err, "Failed to destroy surface object");
    } else {
        SUCCEED() << "Surface object creation not supported, skipping test";
    }
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free CUDA array");
}
