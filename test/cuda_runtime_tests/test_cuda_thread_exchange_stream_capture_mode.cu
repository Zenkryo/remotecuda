#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaThreadExchangeStreamCaptureMode){
    hipStreamCaptureMode mode = hipStreamCaptureModeGlobal;
    hipError_t err = hipThreadExchangeStreamCaptureMode(&mode);
    if(err != hipErrorNotSupported) {
        CHECK_CUDA_ERROR(err, "Failed to exchange stream capture mode");
    }
}
