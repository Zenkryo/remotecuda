#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaEventDestroy){
    hipEvent_t event;
    hipError_t err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
}
