#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemset3D){
    hipPitchedPtr devPtr;
    hipExtent extent = make_hipExtent(32, 32, 32);
    hipError_t err = hipMalloc3D(&devPtr, extent);
    CHECK_CUDA_ERROR(err, "Failed to allocate 3D device memory");
    err = hipMemset3D(devPtr, 0x42, extent);
    CHECK_CUDA_ERROR(err, "Failed to set 3D device memory");
    err = hipFree(devPtr.ptr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
}
