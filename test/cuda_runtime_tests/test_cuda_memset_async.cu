#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemsetAsync){
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    CHECK_CUDA_ERROR(err, "Failed to create stream");
    void *devPtr;
    err = hipMalloc(&devPtr, 1024);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    err = hipMemsetAsync(devPtr, 0x42, 1024, stream);
    CHECK_CUDA_ERROR(err, "Failed to set device memory asynchronously");
    size_t pitch;
    void *devPtr2D;
    err = hipMallocPitch(&devPtr2D, &pitch, 32, 32);
    CHECK_CUDA_ERROR(err, "Failed to allocate pitched device memory");
    err = hipMemset2DAsync(devPtr2D, pitch, 0x42, 32, 32, stream);
    CHECK_CUDA_ERROR(err, "Failed to set 2D device memory asynchronously");
    hipPitchedPtr devPtr3D;
    hipExtent extent = make_hipExtent(32, 32, 32);
    err = hipMalloc3D(&devPtr3D, extent);
    CHECK_CUDA_ERROR(err, "Failed to allocate 3D device memory");
    err = hipMemset3DAsync(devPtr3D, 0x42, extent, stream);
    CHECK_CUDA_ERROR(err, "Failed to set 3D device memory asynchronously");
    err = hipStreamSynchronize(stream);
    CHECK_CUDA_ERROR(err, "Failed to synchronize stream");
    err = hipStreamDestroy(stream);
    CHECK_CUDA_ERROR(err, "Failed to destroy stream");
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
    err = hipFree(devPtr2D);
    CHECK_CUDA_ERROR(err, "Failed to free 2D device memory");
    err = hipFree(devPtr3D.ptr);
    CHECK_CUDA_ERROR(err, "Failed to free 3D device memory");
}
