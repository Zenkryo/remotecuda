#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceSynchronize){
    hipError_t err = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(err, "Failed to synchronize device");
}
