#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemRangeGetAttribute){
    const int size = 1024;
    void *devPtr;
    hipError_t err = hipMalloc(&devPtr, size);
    CHECK_CUDA_ERROR(err, "Failed to allocate device memory");
    hipMemRangeAttribute attr = hipMemRangeAttributeReadMostly;
    int value;
    err = hipMemRangeGetAttribute(&value, sizeof(value), attr, devPtr, size);
    if(err == hipErrorNotSupported) {
        SUCCEED() << "Memory range attribute not supported, skipping test";
    } else if(err == hipErrorInvalidValue) {
        SUCCEED() << "Memory range attribute not valid for this memory, skipping test";
    } else {
        CHECK_CUDA_ERROR(err, "Failed to get memory range attribute");
    }
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
}
