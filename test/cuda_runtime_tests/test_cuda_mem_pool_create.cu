#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemPoolCreate){
    hipMemPool_t memPool = nullptr;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0; // Current device
    poolProps.location.type = hipMemLocationTypeDevice;
    hipError_t err = hipMemPoolCreate(&memPool, &poolProps);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pools not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to create memory pool");
    ASSERT_NE(memPool, nullptr);
    err = hipMemPoolDestroy(memPool);
    CHECK_CUDA_ERROR(err, "Failed to destroy memory pool");
}
