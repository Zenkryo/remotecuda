#include "hip/hip_runtime.h"
#include "common.h"

__device__ float d_symbol[1024];

TEST_F(CudaRuntimeApiTest, CudaGraphSymbolOperations) {
    hipError_t err;
    const int N = 1024;
    size_t size = N * sizeof(float);
    float *h_input = (float *)malloc(size);
    float *h_output = (float *)malloc(size);
    for(int i = 0; i < N; i++) {
        h_input[i] = (float)i;
        h_output[i] = 0.0f;
    }
    float *d_buffer;
    err = hipMalloc(&d_buffer, size);
    CHECK_CUDA_ERROR(err, "hipMalloc d_buffer failed");
    int device;
    hipGetDevice(&device);
    int major, minor;
    hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
    hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
    int computeCapability = major * 10 + minor;
    if(computeCapability < 35) {
        fprintf(stderr, "Device compute capability %d.%d is less than 3.5\n", major, minor);
        hipFree(d_buffer);
        free(h_input);
        free(h_output);
        exit(EXIT_FAILURE);
    }
    void *symbol_addr;
    err = hipGetSymbolAddress(&symbol_addr, HIP_SYMBOL(d_symbol));
    CHECK_CUDA_ERROR(err, "hipGetSymbolAddress failed");
    hipGraph_t graph;
    err = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(err, "hipGraphCreate failed");
    hipGraphNode_t memcpyToSymbolNode, memcpyFromSymbolNode;
    err = hipGraphAddMemcpyNodeToSymbol(&memcpyToSymbolNode, graph, NULL, 0, HIP_SYMBOL(d_symbol), h_input, size, 0, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR(err, "hipGraphAddMemcpyNodeToSymbol failed");
    err = hipGraphAddMemcpyNodeFromSymbol(&memcpyFromSymbolNode, graph, &memcpyToSymbolNode, 1, d_buffer, HIP_SYMBOL(d_symbol), size, 0, hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR(err, "hipGraphAddMemcpyNodeFromSymbol failed");
    hipGraphExec_t graphExec;
    err = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    CHECK_CUDA_ERROR(err, "hipGraphInstantiate failed");
    err = hipGraphLaunch(graphExec, 0);
    CHECK_CUDA_ERROR(err, "hipGraphLaunch failed");
    err = hipMemcpy(h_output, d_buffer, size, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(err, "hipMemcpy failed");
    for(int i = 0; i < N; i++) {
        if(fabs(h_input[i] - h_output[i]) > 1e-5) {
            fprintf(stderr, "Verification failed at index %d: expected %f, got %f\n", i, h_input[i], h_output[i]);
            hipGraphExecDestroy(graphExec);
            hipGraphDestroy(graph);
            hipFree(d_buffer);
            free(h_input);
            free(h_output);
            exit(EXIT_FAILURE);
        }
    }
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipFree(d_buffer);
    free(h_input);
    free(h_output);
}
