#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceMemPool){
    hipMemPool_t defaultPool;
    hipError_t err = hipDeviceGetDefaultMemPool(&defaultPool, 0);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pools not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to get default memory pool");
    ASSERT_NE(defaultPool, nullptr);
    hipMemPool_t newPool = nullptr;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0;
    poolProps.location.type = hipMemLocationTypeDevice;
    err = hipMemPoolCreate(&newPool, &poolProps);
    CHECK_CUDA_ERROR(err, "Failed to create memory pool");
    ASSERT_NE(newPool, nullptr);
    err = hipDeviceSetMemPool(0, newPool);
    CHECK_CUDA_ERROR(err, "Failed to set memory pool");
    hipMemPool_t currentPool;
    err = hipDeviceGetMemPool(&currentPool, 0);
    CHECK_CUDA_ERROR(err, "Failed to get current memory pool");
    ASSERT_EQ(currentPool, newPool) << "Memory pool not set correctly";
    err = hipDeviceSetMemPool(0, defaultPool);
    CHECK_CUDA_ERROR(err, "Failed to restore default memory pool");
    err = hipMemPoolDestroy(newPool);
    CHECK_CUDA_ERROR(err, "Failed to destroy memory pool");
}
