#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaEventCreate){
    hipEvent_t event;
    hipError_t err;
    err = hipEventCreate(&event);
    CHECK_CUDA_ERROR(err, "Failed to create event");
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
    err = hipEventCreateWithFlags(&event, hipEventDisableTiming);
    CHECK_CUDA_ERROR(err, "Failed to create event with flags");
    err = hipEventDestroy(event);
    CHECK_CUDA_ERROR(err, "Failed to destroy event");
}
