#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemPoolAttributes){
    hipMemPool_t memPool = nullptr;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0; // Current device
    poolProps.location.type = hipMemLocationTypeDevice;
    hipError_t err = hipMemPoolCreate(&memPool, &poolProps);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pools not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to create memory pool");
    uint64_t releaseThreshold = 1024 * 1024; // 1MB
    err = hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &releaseThreshold);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pool attributes not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to set release threshold");
    uint64_t retrievedThreshold;
    err = hipMemPoolGetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &retrievedThreshold);
    CHECK_CUDA_ERROR(err, "Failed to get release threshold");
    ASSERT_EQ(retrievedThreshold, releaseThreshold) << "Release threshold not set correctly";
    if(memPool != nullptr) {
        hipMemPoolDestroy(memPool);
    }
}
