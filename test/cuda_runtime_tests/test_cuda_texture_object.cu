#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaTextureObject){
    const int width = 32;
    const int height = 32;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray_t array;
    hipError_t err = hipMallocArray(&array, &channelDesc, width, height);
    CHECK_CUDA_ERROR(err, "Failed to allocate CUDA array");
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = array;
    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    hipTextureObject_t texObj;
    err = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    if(err == hipSuccess) {
        hipResourceDesc retrievedResDesc;
        err = hipGetTextureObjectResourceDesc(&retrievedResDesc, texObj);
        CHECK_CUDA_ERROR(err, "Failed to get texture object resource descriptor");
        hipTextureDesc retrievedTexDesc;
        err = hipGetTextureObjectTextureDesc(&retrievedTexDesc, texObj);
        CHECK_CUDA_ERROR(err, "Failed to get texture object texture descriptor");
        err = hipDestroyTextureObject(texObj);
        CHECK_CUDA_ERROR(err, "Failed to destroy texture object");
    } else {
        SUCCEED() << "Texture object creation not supported, skipping test";
    }
    err = hipFreeArray(array);
    CHECK_CUDA_ERROR(err, "Failed to free CUDA array");
}
