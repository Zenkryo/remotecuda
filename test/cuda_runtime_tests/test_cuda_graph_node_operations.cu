#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaGraphNodeOperations){
    hipGraph_t parentGraph;
    hipError_t err = hipGraphCreate(&parentGraph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create parent graph");
    hipGraph_t childGraph;
    err = hipGraphCreate(&childGraph, 0);
    CHECK_CUDA_ERROR(err, "Failed to create child graph");
    hipGraphNode_t childNode;
    void *kernelArgs[] = {NULL};
    hipKernelNodeParams kernelParams = {0};
    kernelParams.func = (void *)test_kernel;
    kernelParams.gridDim = dim3(1, 1, 1);
    kernelParams.blockDim = dim3(1, 1, 1);
    kernelParams.sharedMemBytes = 0;
    kernelParams.kernelParams = kernelArgs;
    kernelParams.extra = NULL;
    err = hipGraphAddKernelNode(&childNode, childGraph, NULL, 0, &kernelParams);
    CHECK_CUDA_ERROR(err, "Failed to add kernel node to child graph");
    hipGraphNode_t childGraphNode;
    err = hipGraphAddChildGraphNode(&childGraphNode, parentGraph, NULL, 0, childGraph);
    CHECK_CUDA_ERROR(err, "Failed to add child graph node");
    hipGraphExec_t execGraph;
    err = hipGraphInstantiate(&execGraph, parentGraph, NULL, NULL, 0);
    CHECK_CUDA_ERROR(err, "Failed to instantiate graph");
    err = hipGraphExecChildGraphNodeSetParams(execGraph, childGraphNode, childGraph);
    CHECK_CUDA_ERROR(err, "Failed to set child graph parameters");
    err = hipGraphDestroyNode(childGraphNode);
    CHECK_CUDA_ERROR(err, "Failed to destroy child graph node");
    err = hipGraphExecDestroy(execGraph);
    CHECK_CUDA_ERROR(err, "Failed to destroy executable graph");
    err = hipGraphDestroy(parentGraph);
    CHECK_CUDA_ERROR(err, "Failed to destroy parent graph");
    err = hipGraphDestroy(childGraph);
    CHECK_CUDA_ERROR(err, "Failed to destroy child graph");
}
