#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceCacheConfig){
    hipFuncCache_t currentConfig;
    hipError_t err = hipDeviceGetCacheConfig(&currentConfig);
    CHECK_CUDA_ERROR(err, "Failed to get cache config");
    hipFuncCache_t configs[] = {hipFuncCachePreferNone, hipFuncCachePreferShared, hipFuncCachePreferL1, hipFuncCachePreferEqual};
    for(auto config : configs) {
        err = hipDeviceSetCacheConfig(config);
        CHECK_CUDA_ERROR(err, "Failed to set cache config");
        hipFuncCache_t newConfig;
        err = hipDeviceGetCacheConfig(&newConfig);
        CHECK_CUDA_ERROR(err, "Failed to verify cache config");
        ASSERT_EQ(newConfig, config) << "Cache config not set correctly";
    }
    err = hipDeviceSetCacheConfig(currentConfig);
    CHECK_CUDA_ERROR(err, "Failed to restore cache config");
}
