#include "hip/hip_runtime.h"
#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceGetTexture1DLinearMaxWidth){
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if(err == hipSuccess && deviceCount > 0) {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, 0);
        if(err == hipSuccess && prop.major > 0 && prop.canMapHostMemory) {
            size_t maxWidth;
            hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
            err = hipDeviceGetTexture1DLinearMaxWidth(&maxWidth, &desc, 32);
            if(err == hipSuccess) {
                ASSERT_GT(maxWidth, 0) << "Invalid texture 1D linear max width";
            } else {
                SUCCEED() << "Function not supported on this device, skipping test";
            }
        } else {
            SUCCEED() << "Device not suitable, skipping test";
        }
    } else {
        SUCCEED() << "No devices available, skipping test";
    }
}
