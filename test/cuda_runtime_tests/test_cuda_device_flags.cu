#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceFlags){
    unsigned int flags;
    hipError_t err = hipGetDeviceFlags(&flags);
    if(err == hipSuccess) {
        unsigned int flagCombinations[] = {hipDeviceScheduleAuto, hipDeviceScheduleSpin, hipDeviceScheduleYield, hipDeviceScheduleBlockingSync, hipDeviceMapHost, hipDeviceLmemResizeToMax};
        for(auto newFlags : flagCombinations) {
            err = hipSetDeviceFlags(newFlags);
            if(err == hipSuccess) {
                unsigned int currentFlags;
                err = hipGetDeviceFlags(&currentFlags);
                if(err == hipSuccess) {
                    ASSERT_EQ(currentFlags & newFlags, newFlags) << "Device flags not set correctly";
                }
            } else {
                if(err == hipErrorInvalidValue) {
                    SUCCEED() << "Device flags not supported, skipping test";
                }
            }
        }
        err = hipSetDeviceFlags(flags);
        if(err != hipSuccess) {
            SUCCEED() << "Failed to restore flags, but skipping";
        }
    } else {
        SUCCEED() << "Failed to get device flags, skipping test";
    }
}
