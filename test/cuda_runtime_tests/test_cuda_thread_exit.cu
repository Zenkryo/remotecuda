#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaThreadExit){
    hipError_t err = hipDeviceReset();
    CHECK_CUDA_ERROR(err, "Failed to exit thread");
}
