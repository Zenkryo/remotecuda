#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaThreadCacheConfig){
    hipFuncCache_t currentConfig;
    hipError_t err = hipDeviceGetCacheConfig(&currentConfig);
    CHECK_CUDA_ERROR(err, "Failed to get cache config");
    hipFuncCache_t configs[] = {hipFuncCachePreferNone, hipFuncCachePreferShared, hipFuncCachePreferL1, hipFuncCachePreferEqual};
    for(auto config : configs) {
        err = hipDeviceSetCacheConfig(config);
        CHECK_CUDA_ERROR(err, "Failed to set cache config");
        hipFuncCache_t newConfig;
        err = hipDeviceGetCacheConfig(&newConfig);
        CHECK_CUDA_ERROR(err, "Failed to verify cache config");
        ASSERT_EQ(newConfig, config) << "Cache config not set correctly";
    }
    err = hipDeviceSetCacheConfig(currentConfig);
    CHECK_CUDA_ERROR(err, "Failed to restore cache config");
}
