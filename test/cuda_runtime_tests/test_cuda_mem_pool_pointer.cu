#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaMemPoolPointer){
    hipMemPool_t memPool = nullptr;
    hipMemPoolProps poolProps = {};
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.location.id = 0; // Current device
    poolProps.location.type = hipMemLocationTypeDevice;
    hipError_t err = hipMemPoolCreate(&memPool, &poolProps);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pools not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to create memory pool");
    void *devPtr;
    err = hipMallocFromPoolAsync(&devPtr, 1024, memPool, 0);
    if(err == hipErrorNotSupported) {
        GTEST_SKIP() << "Memory pool allocation not supported on this device";
    }
    CHECK_CUDA_ERROR(err, "Failed to allocate memory from pool");
    hipMemPoolPtrExportData exportData;
    err = hipMemPoolExportPointer(&exportData, devPtr);
    if(err == hipSuccess) {
        void *importedPtr;
        err = hipMemPoolImportPointer(&importedPtr, memPool, &exportData);
        if(err == hipSuccess) {
            ASSERT_EQ(importedPtr, devPtr) << "Imported pointer does not match original pointer";
        }
    }
    err = hipFree(devPtr);
    CHECK_CUDA_ERROR(err, "Failed to free device memory");
    err = hipMemPoolDestroy(memPool);
    CHECK_CUDA_ERROR(err, "Failed to destroy memory pool");
}
