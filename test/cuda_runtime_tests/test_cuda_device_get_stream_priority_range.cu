#include "common.h"

TEST_F(CudaRuntimeApiTest, CudaDeviceGetStreamPriorityRange){
    int leastPriority, greatestPriority;
    hipError_t err = hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
    if(err == hipSuccess) {
        SUCCEED() << "Priority range retrieved successfully";
    } else {
        SUCCEED() << "Function not supported, skipping test";
    }
}
