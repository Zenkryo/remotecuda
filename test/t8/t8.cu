#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(val) checkCudaError((val), #val)
void checkCudaError(hipError_t result, const char *func) {
    if(result != hipSuccess) {
        printf("%s failed with error: %s (%d)\n", func, hipGetErrorString(result), result);
    }
}

int main() {
    hipError_t result;
    void *dptr;
    void *hptr;
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t memcpyNode;

    // Allocate host memory
    hptr = malloc(1024);
    if(!hptr) {
        printf("Failed to allocate host memory\n");
        return -1;
    }

    // Allocate device memory
    result = hipMalloc(&dptr, 1024);
    CHECK_CUDA_ERROR(result);
    if(result != hipSuccess) {
        free(hptr);
        return -1;
    }
    printf("Allocated device pointer: %p\n", dptr);

    // Create a CUDA graph
    result = hipGraphCreate(&graph, 0);
    CHECK_CUDA_ERROR(result);
    if(result != hipSuccess) {
        hipFree(dptr);
        free(hptr);
        return -1;
    }

    // Add memory copy node to graph
    hipMemcpy3DParms copyParams = {};
    copyParams.srcPtr.ptr = hptr;
    copyParams.srcPtr.pitch = 1024;
    copyParams.srcPtr.xsize = 1024;
    copyParams.srcPtr.ysize = 1;
    copyParams.dstPtr.ptr = dptr;
    copyParams.dstPtr.pitch = 1024;
    copyParams.dstPtr.xsize = 1024;
    copyParams.dstPtr.ysize = 1;
    copyParams.extent.width = 1024;
    copyParams.extent.height = 1;
    copyParams.extent.depth = 1;
    copyParams.kind = hipMemcpyHostToDevice;

    result = hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &copyParams);
    CHECK_CUDA_ERROR(result);
    if(result != hipSuccess) {
        hipFree(dptr);
        free(hptr);
        hipGraphDestroy(graph);
        return -1;
    }
    printf("Added memory copy node to graph\n");

    // Instantiate the graph
    result = hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    CHECK_CUDA_ERROR(result);
    if(result != hipSuccess) {
        hipFree(dptr);
        free(hptr);
        hipGraphDestroy(graph);
        return -1;
    }
    printf("Instantiated graph\n");

    // Launch the graph
    result = hipGraphLaunch(graphExec, NULL);
    CHECK_CUDA_ERROR(result);
    if(result != hipSuccess) {
        hipFree(dptr);
        free(hptr);
        hipGraphExecDestroy(graphExec);
        hipGraphDestroy(graph);
        return -1;
    }
    printf("Launched graph\n");

    // Synchronize to ensure graph execution is complete
    result = hipDeviceSynchronize();
    CHECK_CUDA_ERROR(result);
    if(result != hipSuccess) {
        hipFree(dptr);
        free(hptr);
        hipGraphExecDestroy(graphExec);
        hipGraphDestroy(graph);
        return -1;
    }
    printf("Graph execution completed\n");

    // Cleanup
    hipFree(dptr);
    free(hptr);
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);

    return 0;
}
