#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
// Macro for checking CUDA errors
#define CUDA_CHECK(call)                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                       \
    do {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                       \
        hipError_t err = call;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                \
        if(err != hipSuccess) {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                               \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err));                                                                                                                                                                                                                                                                                                                                                                                                                 \
            exit(EXIT_FAILURE);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                \
        }                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                      \
    } while(0)

int main() {
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    printf("Found %d CUDA device(s)\n", deviceCount);

    for(int device = 0; device < deviceCount; ++device) {
        hipSetDevice(device);

        // Get device attributes
        int computeCapabilityMajor;
        int computeCapabilityMinor;
        int maxThreadsPerBlock;
        size_t totalGlobalMem;
        int sharedMemPerBlock;
        int maxThreadsPerMultiProcessor;
        int multiProcessorCount;
        int maxGridSize[3];
        int maxThreadsDim[3];
        int warpSize;

        CUDA_CHECK(hipDeviceGetAttribute(&computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, device));
        CUDA_CHECK(hipDeviceGetAttribute(&computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device));
        CUDA_CHECK(hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, device));
        CUDA_CHECK(hipDeviceGetAttribute(&multiProcessorCount, hipDeviceAttributeMultiprocessorCount, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxThreadsPerMultiProcessor, hipDeviceAttributeMaxThreadsPerMultiProcessor, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxGridSize[0], hipDeviceAttributeMaxGridDimX, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxGridSize[1], hipDeviceAttributeMaxGridDimY, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxGridSize[2], hipDeviceAttributeMaxGridDimZ, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxThreadsDim[0], hipDeviceAttributeMaxBlockDimX, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxThreadsDim[1], hipDeviceAttributeMaxBlockDimY, device));
        CUDA_CHECK(hipDeviceGetAttribute(&maxThreadsDim[2], hipDeviceAttributeMaxBlockDimZ, device));
        CUDA_CHECK(hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device));

        hipDeviceProp_t prop;
        CUDA_CHECK(hipGetDeviceProperties(&prop, device));
        totalGlobalMem = prop.totalGlobalMem;

        printf("\nDevice %d:\n", device);
        printf("  Compute Capability: %d.%d\n", computeCapabilityMajor, computeCapabilityMinor);
        printf("  Device Name: %s\n", prop.name);
        printf("  Total Global Memory: %lu bytes\n", (unsigned long)totalGlobalMem);
        printf("  Shared Memory per Block: %lu bytes\n", (unsigned long)sharedMemPerBlock);
        printf("  Max Threads per Block: %d\n", maxThreadsPerBlock);
        printf("  Warp Size: %d\n", warpSize);
        printf("  Number of Multiprocessors: %d\n", multiProcessorCount);
        printf("  Max Threads per Multiprocessor: %d\n", maxThreadsPerMultiProcessor);
        printf("  Max Grid Size: [%d, %d, %d]\n", maxGridSize[0], maxGridSize[1], maxGridSize[2]);
        printf("  Max Thread Block Size: [%d, %d, %d]\n", maxThreadsDim[0], maxThreadsDim[1], maxThreadsDim[2]);
        printf("  Clock Rate: %.2f GHz\n", prop.clockRate * 1e-6f);
        printf("  Memory Clock Rate: %.2f GHz\n", prop.memoryClockRate * 1e-6f);
        printf("  Memory Bus Width: %d bits\n", prop.memoryBusWidth);
        printf("  L2 Cache Size: %d bytes\n", prop.l2CacheSize);

        // Get default memory pool
        hipMemPool_t memPool;
        hipError_t poolErr = hipDeviceGetDefaultMemPool(&memPool, device);
        if(poolErr == hipSuccess) {
            printf("  Default Memory Pool retrieved successfully\n");
        } else {
            printf("  Warning: Default Memory Pool not supported on this device\n");
        }

        // Set and get graph memory attributes
        uint64_t poolLowWatermark = 1024 * 1024 * 512; // 512MB
        hipError_t attrErr = hipDeviceSetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &poolLowWatermark);

        if(attrErr == hipSuccess) {
            uint64_t retrievedLowWatermark;
            CUDA_CHECK(hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &retrievedLowWatermark));

            printf("  Graph Memory Low Watermark: Set to %lu bytes, Retrieved %lu bytes\n", (unsigned long)poolLowWatermark, (unsigned long)retrievedLowWatermark);

            // Get used and reserved memory attributes
            uint64_t usedMem;
            uint64_t reservedMem;
            CUDA_CHECK(hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrUsedMemCurrent, &usedMem));
            CUDA_CHECK(hipDeviceGetGraphMemAttribute(device, hipGraphMemAttrReservedMemCurrent, &reservedMem));

            printf("  Current Used Graph Memory: %lu bytes\n", (unsigned long)usedMem);
            printf("  Current Reserved Graph Memory: %lu bytes\n", (unsigned long)reservedMem);
        } else {
            printf("  Warning: Graph Memory Attributes not supported on this device\n");
        }
    }

    // Reset device
    CUDA_CHECK(hipDeviceReset());
    return 0;
}
