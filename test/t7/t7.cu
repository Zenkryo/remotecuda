#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1024
#define BLOCK_SIZE 16

// CUDA kernel for matrix addition
__global__ void matrixAdd(float *A, float *B, float *C, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n && j < n) {
        C[i * n + j] = A[i * n + j] + B[i * n + j];
    }
}

int main() {
    float *h_A, *h_B, *h_C; // Host matrices
    float *d_A, *d_B, *d_C; // Device matrices
    size_t size = N * N * sizeof(float);
    
    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize host matrices
    for(int i = 0; i < N * N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }
    
    // Register host memory with CUDA
    hipHostRegister(h_A, size, hipHostRegisterDefault);
    hipHostRegister(h_B, size, hipHostRegisterDefault);
    hipHostRegister(h_C, size, hipHostRegisterDefault);
    
    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    // Copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // Set up grid and block dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Launch kernel
    matrixAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    
    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
    // Verify result (simple check)
    for(int i = 0; i < 10; i++) {
        int idx = rand() % (N * N);
        float expected = h_A[idx] + h_B[idx];
        if(fabs(h_C[idx] - expected) > 1e-5) {
            printf("Verification failed at index %d!\n", idx);
            break;
        }
    }
    
    // Cleanup
    hipHostUnregister(h_A);
    hipHostUnregister(h_B);
    hipHostUnregister(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    
    printf("Matrix addition completed!\n");
    return 0;
}
