#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(call)                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                 \
    do {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                       \
        hipError_t err = call;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                   \
        if(err != hipSuccess) {                                                                                                                                                                                                                                                                                                                                                                                                                                                                                              \
            const char *errStr;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                \
            hipDrvGetErrorString(err, &errStr);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                    \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, errStr);                                                                                                                                                                                                                                                                                                                                                                                                                                         \
            exit(EXIT_FAILURE);                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                \
        }                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                      \
    } while(0)

int main() {
    // 初始化CUDA驱动API
    CHECK_CUDA_ERROR(hipInit(0));

    // 获取设备
    hipDevice_t device;
    CHECK_CUDA_ERROR(hipDeviceGet(&device, 0));

    // 创建上下文
    hipCtx_t context;
    CHECK_CUDA_ERROR(hipCtxCreate(&context, 0, device));

    // 分配设备内存
    size_t size = 1024 * 1024; // 1MB
    hipDeviceptr_t d_ptr;
    CHECK_CUDA_ERROR(hipMalloc(&d_ptr, size));

    // 检查设备是否支持所需功能
    int major = 0, minor = 0;
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device));
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device));
    printf("Device compute capability: %d.%d\n", major, minor);

    if(major < 6) {
        fprintf(stderr, "This program requires compute capability 6.0 or higher\n");
        CHECK_CUDA_ERROR(hipFree(d_ptr));
        CHECK_CUDA_ERROR(hipCtxDestroy(context));
        return 1;
    }

    // 准备查询属性
    hipMemRangeAttribute attr = hipMemRangeAttributePreferredLocation;
    size_t data_size = sizeof(hipMemLocation);
    hipMemLocation pref_loc;
    void *data = &pref_loc;

    // 正确调用hipMemRangeGetAttributes
    CHECK_CUDA_ERROR(hipMemRangeGetAttributes(&data,      // void** 指针数组
                                             &data_size, // size_t* 大小数组
                                             &attr,      // hipMemRangeAttribute* 属性数组
                                             1,          // 属性数量
                                             d_ptr,      // 起始地址
                                             size        // 范围大小
                                             ));

    // 打印结果
    printf("Memory range preferred location:\n");
    printf("  Type: %d\n", pref_loc.type);
    printf("  ID: %d\n", pref_loc.id);

    // 清理资源
    CHECK_CUDA_ERROR(hipFree(d_ptr));
    CHECK_CUDA_ERROR(hipCtxDestroy(context));

    return 0;
}
